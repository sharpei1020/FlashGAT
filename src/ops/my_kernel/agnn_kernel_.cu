#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include "agnn.cuh"

#define FULL_MASK 0xffffffff

__global__ void agnn_kernel_16x8_32(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint16_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[2][8];
    __shared__ float dense_X[2][8][32];
    __shared__ float softmax[2][2][16];

    uint32_t D[8], A[2], B[4], E[1] = {0xeeee4444};
    float C[8] = {0.f};
    for (int i=0; i<8; i++) {
        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(D[i]) : "f"(
            x[min(bid*16+(threadIdx.x>>2), node_num-1)*32+i*4+(lane_id&3)]/x_norm[min(bid*16+(threadIdx.x>>2), node_num-1)]));
    }
    int i = block_start;
    int cur_addr;
    SparseAToX_idx[i&1][(threadIdx.x>>3)] = SparseAToX[i*8+(threadIdx.x>>3)];
    cur_addr = __cvta_generic_to_shared(&dense_X[i&1][threadIdx.x>>3][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
    asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i&1][threadIdx.x>>3], node_num-1)*32+(lane_id&7)*4]));
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    softmax[i&1][0][threadIdx.x>>2] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>2] = 0.f;
    __syncthreads();
    for (; i<(block_end-1); i++) {
        SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)] = SparseAToX[(i+1)*8+(threadIdx.x>>3)];
        cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)&1][threadIdx.x>>3][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)&1][threadIdx.x>>3], node_num-1)*32+(lane_id&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
        // SDDMM
        float alpha[4] = {0.f};
        for (int k = 0; k < 2; k++) {
            for (int l = 0; l < 4; l++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                    dense_X[i&1][lane_id>>2][((lane_id&1)*8+((lane_id&2)>>1)+l*2+k*16+(lane_id>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][lane_id>>2], node_num-1)]));
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum;
            int row_id = (lane_id>>2)+warp_id*8;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 2; k++) {
                int col_id = (lane_id&3)*2+k;
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[k] = min(-1.f+2*mask[k], alpha[k]+alpha[k+2])*beta[0];
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][(lane_id>>2)+warp_id*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+warp_id*8] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 2; k++) {
                alpha[k] = mask[k] * __expf(alpha[k]-alpha_max);
                alpha_sum += alpha[k];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][(lane_id>>2)+warp_id*8] * __expf(softmax[i&1][0][(lane_id>>2)+warp_id*8] - alpha_max);
            softmax[(i+1)&1][1][(lane_id>>2)+warp_id*8] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 2; k++) {
                alpha[k] *= rcp;
            }
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(lane_id&3)*2+j+warp_id*8] - softmax[(i+1)&1][0][(lane_id&3)*2+j+warp_id*8])
                        *(softmax[i&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int l = 0; l < 4; l++) {
                    // if (SparseAToX_idx[i&1][(lane_id&3)*2+(l>>1)] < node_num)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][(lane_id&3)*2+(l>>1)][((lane_id>>2)+(l>>1)*4+((l&1)+(lane_id&3))*8+j*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                    : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "r"(A[0]), "r"(A[1]), 
                    "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-1)) {
        // SDDMM
        float alpha[4] = {0.f};
        for (int k = 0; k < 2; k++) {
            for (int l = 0; l < 4; l++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                    dense_X[i&1][lane_id>>2][((lane_id&1)*8+((lane_id&2)>>1)+l*2+k*16+(lane_id>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][lane_id>>2], node_num-1)]));
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum;
            int row_id = (lane_id>>2)+warp_id*8;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 2; k++) {
                int col_id = (lane_id&3)*2+k;
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[k] = min(-1.f+2*mask[k], alpha[k]+alpha[k+2])*beta[0];
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][(lane_id>>2)+warp_id*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+warp_id*8] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 2; k++) {
                alpha[k] = mask[k] * __expf(alpha[k]-alpha_max);
                alpha_sum += alpha[k];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][(lane_id>>2)+warp_id*8] * __expf(softmax[i&1][0][(lane_id>>2)+warp_id*8] - alpha_max);
            softmax[(i+1)&1][1][(lane_id>>2)+warp_id*8] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 2; k++) {
                alpha[k] *= rcp;
            }
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(lane_id&3)*2+j+warp_id*8] - softmax[(i+1)&1][0][(lane_id&3)*2+j+warp_id*8])
                        *(softmax[i&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int l = 0; l < 4; l++) {
                    // if (SparseAToX_idx[i&1][(lane_id&3)*2+(l>>1)] < node_num)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][(lane_id&3)*2+(l>>1)][((lane_id>>2)+(l>>1)*4+((l&1)+(lane_id&3))*8+j*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                    : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "r"(A[0]), "r"(A[1]), 
                    "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
            }
        }
    }
    for (int j = 0; j < 8; j++)
        if (bid*16+(lane_id&3)*2+(j&1)+warp_id*8<node_num)
            output[(bid*16+(lane_id&3)*2+(j&1)+warp_id*8)*32+(j>>1)*8+(lane_id>>2)] = C[j];
}

__global__ void agnn_kernel_16x8_32_3(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint16_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[3][8];
    __shared__ float dense_X[3][8][32];
    __shared__ float softmax[2][2][16];
    __shared__ float sparse_A[2][16][8];

    int lane_front = lane_id>>2;
    int lane_end = lane_id&3;

    uint32_t D[8], A[4], B[4];
    float C[8] = {0.f};
    for (int i=0; i<8; i++) {
        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(D[i]) : "f"(
            x[min(bid*16+(threadIdx.x>>2), node_num-1)*32+i*4+lane_end]/x_norm[min(bid*16+(threadIdx.x>>2), node_num-1)]));
    }
    int i = block_start;
    SparseAToX_idx[i%3][threadIdx.x>>3] = SparseAToX[i*8+(threadIdx.x>>3)];
    int cur_addr = __cvta_generic_to_shared(&dense_X[i%3][threadIdx.x>>3][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
    asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i%3][threadIdx.x>>3], node_num-1)*32+(lane_id&7)*4]));
    asm volatile("cp.async.commit_group;\n"::);
    softmax[i&1][0][threadIdx.x>>2] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>2] = 0.f;
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    if ((i+1) < block_end) {
        SparseAToX_idx[(i+1)%3][threadIdx.x>>3] = SparseAToX[(i+1)*8+(threadIdx.x>>3)];
        cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)%3][threadIdx.x>>3][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)%3][threadIdx.x>>3], node_num-1)*32+(lane_id&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
    }
    float alpha[4] = {0.f};
    uint32_t E[1] = {0xeeee4444};
    for (int j = 0; j < 2; j++) {
        for (int k = 0; k < 4; k++) {
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense_X[i%3][lane_front][(j*16+lane_front*4+k*2+(lane_end>>1)+(lane_end&1)*8)&31]/x_norm[min(SparseAToX_idx[i%3][lane_front], node_num-1)]));
        }
        asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
            "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
            : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
            : "r"(D[j*4]), "r"(D[j*4+2]), "r"(D[j*4+1]), "r"(D[j*4+3]), 
            "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
            "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
    }
    for (int j = 0; j < 2; j++) {
        sparse_A[i&1][lane_front+warp_id*8][(lane_end*2+(j&1)+(lane_front>>2))&7] = alpha[j]+alpha[j+2];
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    for (; i<(block_end-2); i++) {
        SparseAToX_idx[(i+2)%3][threadIdx.x>>3] = SparseAToX[(i+2)*8+(threadIdx.x>>3)];
        cur_addr = __cvta_generic_to_shared(&dense_X[(i+2)%3][threadIdx.x>>3][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+2)%3][threadIdx.x>>3], node_num-1)*32+(lane_id&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
        float alpha[4] = {0.f};
        for (int j = 0; j < 2; j++) {
            for (int k = 0; k < 4; k++) {
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense_X[(i+1)%3][lane_front][(j*16+lane_front*4+k*2+(lane_end>>1)+(lane_end&1)*8)&31]/x_norm[min(SparseAToX_idx[(i+1)%3][lane_front], node_num-1)]));
            }
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[j*4]), "r"(D[j*4+2]), "r"(D[j*4+1]), "r"(D[j*4+3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        }
        for (int j = 0; j < 2; j++) {
            sparse_A[(i+1)&1][lane_front+warp_id*8][(lane_end*2+(j&1)+(lane_front>>2))&7] = alpha[j]+alpha[j+2];
        }
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max[2], alpha_sum[2];
            for (int j = 0; j < 2; j++) {
                int row_id = lane_front+j*8;
                uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
                for (int k = 0; k < 2; k++) {
                    int col_id = lane_end*2+k;
                    mask[2*j+k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                    alpha[2*j+k] = min(-1.f+2*mask[2*j+k], sparse_A[i&1][lane_front+j*8][(lane_end*2+k+(lane_front>>2))&7])*beta[0];
                }
                alpha_max[j] = max(alpha[j*2], alpha[j*2+1]);
                for (int k = 1; k < 4; k<<=1) {
                    alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
                }
                alpha_max[j] = max(alpha_max[j], softmax[i&1][0][lane_front+j*8]);
                softmax[(i+1)&1][0][lane_front+j*8] = alpha_max[j];
                alpha[j*2] = mask[j*2] * __expf(alpha[j*2]-alpha_max[j]);
                alpha[j*2+1] = mask[j*2+1] * __expf(alpha[j*2+1]-alpha_max[j]);
                alpha_sum[j] = alpha[j*2] + alpha[j*2+1];
                for (int k = 1; k < 4; k<<=1) {
                    alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
                }
                alpha_sum[j] += softmax[i&1][1][lane_front+j*8] * __expf(softmax[i&1][0][lane_front+j*8] - alpha_max[j]);
                softmax[(i+1)&1][1][lane_front+j*8] = alpha_sum[j];
                float rcp = 1.f/(alpha_sum[j]+1e-16f);
                alpha[j*2] *= rcp;
                alpha[j*2+1] *= rcp;
            }
            for (int j = 0; j < 4; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][lane_front+j*8] - softmax[(i+1)&1][0][lane_front+j*8])
                        *(softmax[i&1][1][lane_front+j*8]+1e-16f)/(softmax[(i+1)&1][1][lane_front+j*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j*2+(k&2)*2+(k&1)] *= update;
            }
            for (int k = 0; k < 2; k++) {
                for (int j = 0; j < 2; j++) {
                    // if (SparseAToX_idx[i%3][lane_end*2+j] < node_num) 
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense_X[i%3][lane_end*2+j][(lane_front+(lane_end*2+j)*4+k*8+warp_id*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));            
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[k*4]), "=f"(C[k*4+1]), "=f"(C[k*4+2]), "=f"(C[k*4+3])
                    : "r"(A[0]), "r"(A[2]), "r"(A[1]), "r"(A[3]), 
                    "r"(B[0]), "r"(B[1]), 
                    "f"(C[k*4]), "f"(C[k*4+1]), "f"(C[k*4+2]), "f"(C[k*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-2))
    {
        float alpha[4] = {0.f};
        for (int j = 0; j < 2; j++) {
            for (int k = 0; k < 4; k++) {
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense_X[(i+1)%3][lane_front][(j*16+lane_front*4+k*2+(lane_end>>1)+(lane_end&1)*8)&31]/x_norm[min(SparseAToX_idx[(i+1)%3][lane_front], node_num-1)]));
            }
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[j*4]), "r"(D[j*4+2]), "r"(D[j*4+1]), "r"(D[j*4+3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        }
        for (int j = 0; j < 2; j++) {
            sparse_A[(i+1)&1][lane_front+warp_id*8][(lane_end*2+(j&1)+(lane_front>>2))&7] = alpha[j]+alpha[j+2];
        }
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max[2], alpha_sum[2];
            for (int j = 0; j < 2; j++) {
                int row_id = lane_front+j*8;
                uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
                for (int k = 0; k < 2; k++) {
                    int col_id = lane_end*2+k;
                    mask[2*j+k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                    alpha[2*j+k] = min(-1.f+2*mask[2*j+k], sparse_A[i&1][lane_front+j*8][(lane_end*2+k+(lane_front>>2))&7])*beta[0];
                }
                alpha_max[j] = max(alpha[j*2], alpha[j*2+1]);
                for (int k = 1; k < 4; k<<=1) {
                    alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
                }
                alpha_max[j] = max(alpha_max[j], softmax[i&1][0][lane_front+j*8]);
                softmax[(i+1)&1][0][lane_front+j*8] = alpha_max[j];
                alpha[j*2] = mask[j*2] * __expf(alpha[j*2]-alpha_max[j]);
                alpha[j*2+1] = mask[j*2+1] * __expf(alpha[j*2+1]-alpha_max[j]);
                alpha_sum[j] = alpha[j*2] + alpha[j*2+1];
                for (int k = 1; k < 4; k<<=1) {
                    alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
                }
                alpha_sum[j] += softmax[i&1][1][lane_front+j*8] * __expf(softmax[i&1][0][lane_front+j*8] - alpha_max[j]);
                softmax[(i+1)&1][1][lane_front+j*8] = alpha_sum[j];
                float rcp = 1.f/(alpha_sum[j]+1e-16f);
                alpha[j*2] *= rcp;
                alpha[j*2+1] *= rcp;
            }
            for (int j = 0; j < 4; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][lane_front+j*8] - softmax[(i+1)&1][0][lane_front+j*8])
                        *(softmax[i&1][1][lane_front+j*8]+1e-16f)/(softmax[(i+1)&1][1][lane_front+j*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j*2+(k&2)*2+(k&1)] *= update;
            }
            for (int k = 0; k < 2; k++) {
                for (int j = 0; j < 2; j++) {
                    // if (SparseAToX_idx[i%3][lane_end*2+j] < node_num) 
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense_X[i%3][lane_end*2+j][(lane_front+(lane_end*2+j)*4+k*8+warp_id*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));            
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[k*4]), "=f"(C[k*4+1]), "=f"(C[k*4+2]), "=f"(C[k*4+3])
                    : "r"(A[0]), "r"(A[2]), "r"(A[1]), "r"(A[3]), 
                    "r"(B[0]), "r"(B[1]), 
                    "f"(C[k*4]), "f"(C[k*4+1]), "f"(C[k*4+2]), "f"(C[k*4+3]));
            }
        }
        i++;
        __syncthreads();
    }
    if (i == (block_end-1)) {
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max[2], alpha_sum[2];
            for (int j = 0; j < 2; j++) {
                int row_id = lane_front+j*8;
                uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
                for (int k = 0; k < 2; k++) {
                    int col_id = lane_end*2+k;
                    mask[2*j+k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                    alpha[2*j+k] = min(-1.f+2*mask[2*j+k], sparse_A[i&1][lane_front+j*8][(lane_end*2+k+(lane_front>>2))&7])*beta[0];
                }
                alpha_max[j] = max(alpha[j*2], alpha[j*2+1]);
                for (int k = 1; k < 4; k<<=1) {
                    alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
                }
                alpha_max[j] = max(alpha_max[j], softmax[i&1][0][lane_front+j*8]);
                softmax[(i+1)&1][0][lane_front+j*8] = alpha_max[j];
                alpha[j*2] = mask[j*2] * __expf(alpha[j*2]-alpha_max[j]);
                alpha[j*2+1] = mask[j*2+1] * __expf(alpha[j*2+1]-alpha_max[j]);
                alpha_sum[j] = alpha[j*2] + alpha[j*2+1];
                for (int k = 1; k < 4; k<<=1) {
                    alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
                }
                alpha_sum[j] += softmax[i&1][1][lane_front+j*8] * __expf(softmax[i&1][0][lane_front+j*8] - alpha_max[j]);
                softmax[(i+1)&1][1][lane_front+j*8] = alpha_sum[j];
                float rcp = 1.f/(alpha_sum[j]+1e-16f);
                alpha[j*2] *= rcp;
                alpha[j*2+1] *= rcp;
            }
            for (int j = 0; j < 4; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][lane_front+j*8] - softmax[(i+1)&1][0][lane_front+j*8])
                        *(softmax[i&1][1][lane_front+j*8]+1e-16f)/(softmax[(i+1)&1][1][lane_front+j*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j*2+(k&2)*2+(k&1)] *= update;
            }
            for (int k = 0; k < 2; k++) {
                for (int j = 0; j < 2; j++) {
                    // if (SparseAToX_idx[i%3][lane_end*2+j] < node_num) 
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense_X[i%3][lane_end*2+j][(lane_front+(lane_end*2+j)*4+k*8+warp_id*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));            
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[k*4]), "=f"(C[k*4+1]), "=f"(C[k*4+2]), "=f"(C[k*4+3])
                    : "r"(A[0]), "r"(A[2]), "r"(A[1]), "r"(A[3]), 
                    "r"(B[0]), "r"(B[1]), 
                    "f"(C[k*4]), "f"(C[k*4+1]), "f"(C[k*4+2]), "f"(C[k*4+3]));
            }
        }
    }
    for (int j = 0; j < 4; j++)
        if (bid*16+lane_front+(j&1)*8<node_num) 
            *(float2*)(&output[(bid*16+lane_front+(j&1)*8)*32+warp_id*16+(j&2)*4+lane_end*2]) = *(float2*)(&C[j*2]);
}

__global__ void agnn_kernel_16x16_32(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint16_t* __restrict__ BitColMask,
    const uint16_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[2][16];
    __shared__ float dense_X[2][16][32];
    __shared__ float softmax[2][2][16];

    uint32_t D[8], A[4], B[4], E[1] = {0xeeee4444};
    float C[8] = {0.f};
    for (int i=0; i<8; i++) {
        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(D[i]) : "f"(
            x[min(bid*16+(threadIdx.x>>2), node_num-1)*32+i*4+(lane_id&3)]/x_norm[min(bid*16+(threadIdx.x>>2), node_num-1)]));
    }
    int i = block_start;
    int cur_addr;
    for (int j = 0; j < 2; j++) {
        SparseAToX_idx[i&1][(threadIdx.x>>3)+j*8] = SparseAToX[i*16+(threadIdx.x>>3)+j*8];
        cur_addr = __cvta_generic_to_shared(&dense_X[i&1][(threadIdx.x>>3)+j*8][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i&1][(threadIdx.x>>3)+j*8], node_num-1)*32+(lane_id&7)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    softmax[i&1][0][threadIdx.x>>2] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>2] = 0.f;
    __syncthreads();
    for (; i<(block_end-1); i++) {
        for (int j = 0; j < 2; j++) {
            SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)+j*8] = SparseAToX[(i+1)*16+(threadIdx.x>>3)+j*8];
            cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)&1][(threadIdx.x>>3)+j*8][(((lane_id&7)+(threadIdx.x>>3))*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)+j*8], node_num-1)*32+(lane_id&7)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // SDDMM
        float alpha[8] = {0.f};
        for (int j = 0; j < 2; j++) {
            for (int k = 0; k < 2; k++) {
                for (int l = 0; l < 4; l++)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                        dense_X[i&1][(lane_id>>2)+j*8][((lane_id&1)*8+((lane_id&2)>>1)+l*2+k*16+(lane_id>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][(lane_id>>2)+j*8], node_num-1)]));
                asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                    "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                    : "=f"(alpha[j*4]), "=f"(alpha[j*4+1]), "=f"(alpha[j*4+2]), "=f"(alpha[j*4+3])
                    : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                    "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "f"(alpha[j*4]), "f"(alpha[j*4+1]), "f"(alpha[j*4+2]), "f"(alpha[j*4+3]), "r"(E[0]));
            }
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max, alpha_sum;
            int row_id = (lane_id>>2)+warp_id*8;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 4; k++) {
                int col_id = (lane_id&3)*2+(k&2)*4+(k&1);
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[(k&2)*2+(k&1)] = min(-1.f+2*mask[k], alpha[(k&2)*2+(k&1)]+alpha[(k&2)*2+(k&1)+2])*beta[0];
            }
            alpha_max = max(max(alpha[0], alpha[1]), max(alpha[4], alpha[5]));
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][(lane_id>>2)+warp_id*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+warp_id*8] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 4; k++) {
                alpha[(k&2)*2+(k&1)] = mask[k] * __expf(alpha[(k&2)*2+(k&1)]-alpha_max);
                alpha_sum += alpha[(k&2)*2+(k&1)];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][(lane_id>>2)+warp_id*8] * __expf(softmax[i&1][0][(lane_id>>2)+warp_id*8] - alpha_max);
            softmax[(i+1)&1][1][(lane_id>>2)+warp_id*8] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 4; k++) {
                alpha[(k&2)*2+(k&1)] *= rcp;
            }
            for (int j = 0; j < 4; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[(j&2)*2+(j&1)]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(lane_id&3)*2+j+warp_id*8] - softmax[(i+1)&1][0][(lane_id&3)*2+j+warp_id*8])
                        *(softmax[i&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int k = 0; k < 2; k++) {
                    for (int l = 0; l < 4; l++) {
                        // if (SparseAToX_idx[i&1][k*8+(lane_id&3)*2+(l>>1)] < node_num)
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][k*8+(lane_id&3)*2+(l>>1)][((lane_id>>2)+(l>>1)*4+((l&1)+(lane_id&3))*8+j*16)&31]));
                        // else
                        //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                    }
                    asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                        : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                        : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                        "r"(A[k*2]), "r"(A[k*2+1]), 
                        "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
                }
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-1)) {
        // SDDMM
        float alpha[8] = {0.f};
        for (int j = 0; j < 2; j++) {
            for (int k = 0; k < 2; k++) {
                for (int l = 0; l < 4; l++)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                        dense_X[i&1][(lane_id>>2)+j*8][((lane_id&1)*8+((lane_id&2)>>1)+l*2+k*16+(lane_id>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][(lane_id>>2)+j*8], node_num-1)]));
                asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                    "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                    : "=f"(alpha[j*4]), "=f"(alpha[j*4+1]), "=f"(alpha[j*4+2]), "=f"(alpha[j*4+3])
                    : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                    "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "f"(alpha[j*4]), "f"(alpha[j*4+1]), "f"(alpha[j*4+2]), "f"(alpha[j*4+3]), "r"(E[0]));
            }
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max, alpha_sum;
            int row_id = (lane_id>>2)+warp_id*8;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 4; k++) {
                int col_id = (lane_id&3)*2+(k&2)*4+(k&1);
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[(k&2)*2+(k&1)] = min(-1.f+2*mask[k], alpha[(k&2)*2+(k&1)]+alpha[(k&2)*2+(k&1)+2])*beta[0];
            }
            alpha_max = max(max(alpha[0], alpha[1]), max(alpha[4], alpha[5]));
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][(lane_id>>2)+warp_id*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+warp_id*8] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 4; k++) {
                alpha[(k&2)*2+(k&1)] = mask[k] * __expf(alpha[(k&2)*2+(k&1)]-alpha_max);
                alpha_sum += alpha[(k&2)*2+(k&1)];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][(lane_id>>2)+warp_id*8] * __expf(softmax[i&1][0][(lane_id>>2)+warp_id*8] - alpha_max);
            softmax[(i+1)&1][1][(lane_id>>2)+warp_id*8] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 4; k++) {
                alpha[(k&2)*2+(k&1)] *= rcp;
            }
            for (int j = 0; j < 4; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[(j&2)*2+(j&1)]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(lane_id&3)*2+j+warp_id*8] - softmax[(i+1)&1][0][(lane_id&3)*2+j+warp_id*8])
                        *(softmax[i&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id&3)*2+j+warp_id*8]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int k = 0; k < 2; k++) {
                    for (int l = 0; l < 4; l++) {
                        // if (SparseAToX_idx[i&1][k*8+(lane_id&3)*2+(l>>1)] < node_num)
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][k*8+(lane_id&3)*2+(l>>1)][((lane_id>>2)+(l>>1)*4+((l&1)+(lane_id&3))*8+j*16)&31]));
                        // else
                        //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                    }
                    asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                        : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                        : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                        "r"(A[k*2]), "r"(A[k*2+1]), 
                        "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
                }
            }
        }
    }
    for (int j = 0; j < 8; j++)
        if (bid*16+(lane_id&3)*2+(j&1)+warp_id*8<node_num)
            output[(bid*16+(lane_id&3)*2+(j&1)+warp_id*8)*32+(j>>1)*8+(lane_id>>2)] = C[j];
}


__global__ void agnn_kernel_8x16_32(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint16_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[2][16];
    __shared__ float dense_X[2][16][32];
    __shared__ float softmax[2][2][8];

    uint32_t D[8], A[4], B[4], E[1] = {0xeeee4444};
    float C[8] = {0.f};
    for (int i=0; i<8; i++) {
        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(D[i]) : "f"(
            x[min(bid*8+(threadIdx.x>>2), node_num-1)*32+i*4+(threadIdx.x&3)]/x_norm[min(bid*8+(threadIdx.x>>2), node_num-1)]));
    }
    int i = block_start;
    int cur_addr;
    for (int j = 0; j < 4; j++) {
        SparseAToX_idx[i&1][(threadIdx.x>>3)+j*4] = SparseAToX[i*16+(threadIdx.x>>3)+j*4];
        cur_addr = __cvta_generic_to_shared(&dense_X[i&1][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i&1][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    softmax[i&1][0][threadIdx.x>>2] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>2] = 0.f;
    __syncthreads();
    for (; i<(block_end-1); i++) {
        for (int j = 0; j < 4; j++) {
            SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)+j*4] = SparseAToX[(i+1)*16+(threadIdx.x>>3)+j*4];
            cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)&1][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // SDDMM
        float alpha[8] = {0.f};
        for (int j = 0; j < 2; j++) {
            for (int k = 0; k < 2; k++) {
                for (int l = 0; l < 4; l++)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                        dense_X[i&1][(threadIdx.x>>2)+j*8][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+l*2+k*16+(threadIdx.x>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][(threadIdx.x>>2)+j*8], node_num-1)]));
                asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                    "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                    : "=f"(alpha[j*4]), "=f"(alpha[j*4+1]), "=f"(alpha[j*4+2]), "=f"(alpha[j*4+3])
                    : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                    "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "f"(alpha[j*4]), "f"(alpha[j*4+1]), "f"(alpha[j*4+2]), "f"(alpha[j*4+3]), "r"(E[0]));
            }
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max, alpha_sum;
            int row_id = threadIdx.x>>2;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 4; k++) {
                int col_id = (threadIdx.x&3)*2+(k&2)*4+(k&1);
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[(k&2)*2+(k&1)] = min(-1.f+2*mask[k], alpha[(k&2)*2+(k&1)]+alpha[(k&2)*2+(k&1)+2])*beta[0];
            }
            alpha_max = max(max(alpha[0], alpha[1]), max(alpha[4], alpha[5]));
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][threadIdx.x>>2]);
            softmax[(i+1)&1][0][threadIdx.x>>2] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 4; k++) {
                alpha[(k&2)*2+(k&1)] = mask[k] * __expf(alpha[(k&2)*2+(k&1)]-alpha_max);
                alpha_sum += alpha[(k&2)*2+(k&1)];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>2] * __expf(softmax[i&1][0][threadIdx.x>>2] - alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>2] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 4; k++) {
                alpha[(k&2)*2+(k&1)] *= rcp;
            }
            for (int j = 0; j < 4; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[(j&2)*2+(j&1)]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j] - softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                        *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int k = 0; k < 2; k++) {
                    for (int l = 0; l < 4; l++) {
                        // if (SparseAToX_idx[i&1][k*8+(threadIdx.x&3)*2+(l>>1)] < node_num)
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][k*8+(threadIdx.x&3)*2+(l>>1)][((threadIdx.x>>2)+(l>>1)*4+((l&1)+(threadIdx.x&3))*8+j*16)&31]));
                        // else
                        //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                    }
                    asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                        : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                        : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                        "r"(A[k*2]), "r"(A[k*2+1]), 
                        "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
                }
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-1)) {
       // SDDMM
       float alpha[8] = {0.f};
       for (int j = 0; j < 2; j++) {
           for (int k = 0; k < 2; k++) {
               for (int l = 0; l < 4; l++)
                   asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                       dense_X[i&1][(threadIdx.x>>2)+j*8][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+l*2+k*16+(threadIdx.x>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][(threadIdx.x>>2)+j*8], node_num-1)]));
               asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                   "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                   : "=f"(alpha[j*4]), "=f"(alpha[j*4+1]), "=f"(alpha[j*4+2]), "=f"(alpha[j*4+3])
                   : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                   "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                   "f"(alpha[j*4]), "f"(alpha[j*4+1]), "f"(alpha[j*4+2]), "f"(alpha[j*4+3]), "r"(E[0]));
           }
       }
       // Softmax
       {
           int row_start = BitMaskRowOffset[i];
           int row_end = BitMaskRowOffset[i+1];
           uint32_t col_mask = BitColMask[i];
           float mask[4], alpha_max, alpha_sum;
           int row_id = threadIdx.x>>2;
           uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
           for (int k = 0; k < 4; k++) {
               int col_id = (threadIdx.x&3)*2+(k&2)*4+(k&1);
               mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
               alpha[(k&2)*2+(k&1)] = min(-1.f+2*mask[k], alpha[(k&2)*2+(k&1)]+alpha[(k&2)*2+(k&1)+2])*beta[0];
           }
           alpha_max = max(max(alpha[0], alpha[1]), max(alpha[4], alpha[5]));
           for (int k = 1; k < 4; k<<=1) {
               alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
           }
           alpha_max = max(alpha_max, softmax[i&1][0][threadIdx.x>>2]);
           softmax[(i+1)&1][0][threadIdx.x>>2] = alpha_max;
           alpha_sum = 0;
           for (int k = 0; k < 4; k++) {
               alpha[(k&2)*2+(k&1)] = mask[k] * __expf(alpha[(k&2)*2+(k&1)]-alpha_max);
               alpha_sum += alpha[(k&2)*2+(k&1)];
           }
           for (int k = 1; k < 4; k<<=1) {
               alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
           }
           alpha_sum += softmax[i&1][1][threadIdx.x>>2] * __expf(softmax[i&1][0][threadIdx.x>>2] - alpha_max);
           softmax[(i+1)&1][1][threadIdx.x>>2] = alpha_sum;
           float rcp = 1.f/(alpha_sum+1e-16f);
           for (int k = 0; k < 4; k++) {
               alpha[(k&2)*2+(k&1)] *= rcp;
           }
           for (int j = 0; j < 4; j++)
               asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[(j&2)*2+(j&1)]));
       }
       // Matmul
       {
           for (int j = 0; j < 2; j++) {
               float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j] - softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                       *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
               for (int k = 0; k < 4; k++) 
                   C[j+k*2] *= update;
           }
           for (int j = 0; j < 2; j++) {
               for (int k = 0; k < 2; k++) {
                   for (int l = 0; l < 4; l++) {
                    //    if (SparseAToX_idx[i&1][k*8+(threadIdx.x&3)*2+(l>>1)] < node_num)
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][k*8+(threadIdx.x&3)*2+(l>>1)][((threadIdx.x>>2)+(l>>1)*4+((l&1)+(threadIdx.x&3))*8+j*16)&31]));
                    //    else
                    //        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                   }
                   asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                       : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                       : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                       "r"(A[k*2]), "r"(A[k*2+1]), 
                       "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
               }
           }
       } 
    }
    for (int j = 0; j < 8; j++)
        if (bid*8+(threadIdx.x&3)*2+(j&1)<node_num)
            output[(bid*8+(threadIdx.x&3)*2+(j&1))*32+(j>>1)*8+(threadIdx.x>>2)] = C[j];
}

__global__ void agnn_kernel_8x8_32_3(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[3][8];
    __shared__ float dense_X[3][8][32];
    __shared__ float softmax[2][2][8];
    __shared__ float sparse_A[2][16][8];

    uint32_t D[4], A[4], B[4], E[1] = {0xeeee4444};
    float C[4];
    float scale = beta[0];
    for (int i=0; i<4; i++) {
        C[i] = x[min(bid*8+((threadIdx.x>>2)&7), node_num-1)*32+i*4+(threadIdx.x>>5)*16]/x_norm[min(bid*8+((threadIdx.x>>2)&7), node_num-1)];
        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(D[i]) : "f"(C[i]));
        C[i] = 0.f;
    }
    int i = block_start;
    SparseAToX_idx[i%3][threadIdx.x>>3] = SparseAToX[i*8+(threadIdx.x>>3)];
    int cur_addr = __cvta_generic_to_shared(&dense_X[i%3][threadIdx.x>>3][(((threadIdx.x&7)+(threadIdx.x>>3))*4)&31]);
    asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i%3][threadIdx.x>>3], node_num-1)*32+(threadIdx.x&7)*4]));
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    if ((i+1)<block_end) {
        SparseAToX_idx[(i+1)%3][threadIdx.x>>3] = SparseAToX[(i+1)*8+(threadIdx.x>>3)];
        cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)%3][threadIdx.x>>3][(((threadIdx.x&7)+(threadIdx.x>>3))*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)%3][threadIdx.x>>3], node_num-1)*32+(threadIdx.x&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
    }
    {
        float alpha[4] = {0.f};
        for (int k = 0; k < 4; k++)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(
                dense_X[i%3][((threadIdx.x>>2)&7)][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+k*2+4*((threadIdx.x>>2)&7)+16*(threadIdx.x>>5))&31]/x_norm[min(SparseAToX_idx[i%3][((threadIdx.x>>2)&7)], node_num-1)]));
        asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
            "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
            : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
            : "r"(D[0]), "r"(D[2]), "r"(D[1]), "r"(D[3]), 
            "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
            "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        for (int k = 0; k < 2; k++)
            sparse_A[i&1][(threadIdx.x>>3)+8*k][threadIdx.x&7] = alpha[k]+alpha[k+2];
    }
    softmax[i&1][0][threadIdx.x>>3] = -1.0f * scale;
    softmax[i&1][1][threadIdx.x>>3] = 0.f;
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    for (; i<(block_end-2); i++) {
        SparseAToX_idx[(i+2)%3][threadIdx.x>>3] = SparseAToX[(i+2)*8+(threadIdx.x>>3)];
        cur_addr = __cvta_generic_to_shared(&dense_X[(i+2)%3][threadIdx.x>>3][(((threadIdx.x&7)+(threadIdx.x>>3))*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+2)%3][threadIdx.x>>3], node_num-1)*32+(threadIdx.x&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
        {
            float alpha[4] = {0.f};
            for (int k = 0; k < 4; k++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(
                    dense_X[(i+1)%3][((threadIdx.x>>2)&7)][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+k*2+4*((threadIdx.x>>2)&7)+16*(threadIdx.x>>5))&31]/x_norm[min(SparseAToX_idx[(i+1)%3][((threadIdx.x>>2)&7)], node_num-1)]));
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[0]), "r"(D[2]), "r"(D[1]), "r"(D[3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
            for (int k = 0; k < 2; k++)
                sparse_A[(i+1)&1][(threadIdx.x>>3)+k*8][threadIdx.x&7] = alpha[k]+alpha[k+2];
        }
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum, alpha[2];
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<((threadIdx.x>>2)&7))-1))];
            for (int k = 0; k < 2; k++) {
                mask[k] = (float)(((col_mask>>((threadIdx.x>>2)&7))&(row_mask>>((threadIdx.x&3)*2+k)))&1);
                alpha[k] = min(-1.f+2*mask[k], sparse_A[i&1][((threadIdx.x>>3)&3)+k*8][threadIdx.x&7]+sparse_A[i&1][((threadIdx.x>>3)&3)+4+k*8][threadIdx.x&7])*scale;
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][((threadIdx.x>>2)&7)]);
            softmax[(i+1)&1][0][((threadIdx.x>>2)&7)] = alpha_max;
            alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
            alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
            alpha_sum = alpha[0] + alpha[1];
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1);
            }
            alpha_sum += softmax[i&1][1][((threadIdx.x>>2)&7)]*__expf(softmax[i&1][0][((threadIdx.x>>2)&7)]-alpha_max);
            softmax[(i+1)&1][1][((threadIdx.x>>2)&7)] = alpha_sum;
            float rcp = 1.f / (alpha_sum+1e-16f);
            alpha[0] *= rcp;
            alpha[1] *= rcp;
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(alpha[j]));
        }
        for (int j=0; j<2; j++) {
            float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j]-softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                        *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
            C[j] *= update;
            C[j+2] *= update;
        }
        for (int j=0; j<2; j++) {
            // if (SparseAToX_idx[i%3][(threadIdx.x&3)*2+j]<node_num) {
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2]) : "f"(dense_X[i%3][(threadIdx.x&3)*2+j][(((threadIdx.x>>2)&7)+(threadIdx.x>>5)*16+((threadIdx.x&3)*2+j)*4)&31]));
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2+1]) : "f"(dense_X[i%3][(threadIdx.x&3)*2+j][(((threadIdx.x>>2)&7)+(threadIdx.x>>5)*16+8+((threadIdx.x&3)*2+j)*4)&31]));
            // } else {
            //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2]) : "f"(0.f));
            //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2+1]) : "f"(0.f));
            // }
        }
        asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
            : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
            "r"(B[0]), "r"(B[1]), 
            "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-2)) {
        {
            float alpha[4] = {0.f};
            for (int k = 0; k < 4; k++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(
                    dense_X[(i+1)%3][((threadIdx.x>>2)&7)][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+k*2+4*((threadIdx.x>>2)&7)+16*(threadIdx.x>>5))&31]/x_norm[min(SparseAToX_idx[(i+1)%3][((threadIdx.x>>2)&7)], node_num-1)]));
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[0]), "r"(D[2]), "r"(D[1]), "r"(D[3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
            for (int k = 0; k < 2; k++)
                sparse_A[(i+1)&1][(threadIdx.x>>3)+k*8][threadIdx.x&7] = alpha[k]+alpha[k+2];
        }
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum, alpha[2];
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<((threadIdx.x>>2)&7))-1))];
            for (int k = 0; k < 2; k++) {
                mask[k] = (float)(((col_mask>>((threadIdx.x>>2)&7))&(row_mask>>((threadIdx.x&3)*2+k)))&1);
                alpha[k] = min(-1.f+2*mask[k], sparse_A[i&1][((threadIdx.x>>3)&3)+k*8][threadIdx.x&7]+sparse_A[i&1][((threadIdx.x>>3)&3)+4+k*8][threadIdx.x&7])*scale;
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][((threadIdx.x>>2)&7)]);
            softmax[(i+1)&1][0][((threadIdx.x>>2)&7)] = alpha_max;
            alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
            alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
            alpha_sum = alpha[0] + alpha[1];
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1);
            }
            alpha_sum += softmax[i&1][1][((threadIdx.x>>2)&7)]*__expf(softmax[i&1][0][((threadIdx.x>>2)&7)]-alpha_max);
            softmax[(i+1)&1][1][((threadIdx.x>>2)&7)] = alpha_sum;
            float rcp = 1.f / (alpha_sum+1e-16f);
            alpha[0] *= rcp;
            alpha[1] *= rcp;
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(alpha[j]));
        }
        for (int j=0; j<2; j++) {
            float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j]-softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                        *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
            C[j] *= update;
            C[j+2] *= update;
        }
        for (int j=0; j<2; j++) {
            // if (SparseAToX_idx[i%3][(threadIdx.x&3)*2+j]<node_num) {
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2]) : "f"(dense_X[i%3][(threadIdx.x&3)*2+j][(((threadIdx.x>>2)&7)+(threadIdx.x>>5)*16+((threadIdx.x&3)*2+j)*4)&31]));
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2+1]) : "f"(dense_X[i%3][(threadIdx.x&3)*2+j][(((threadIdx.x>>2)&7)+(threadIdx.x>>5)*16+8+((threadIdx.x&3)*2+j)*4)&31]));
            // } else {
            //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2]) : "f"(0.f));
            //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2+1]) : "f"(0.f));
            // }
        }
        asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
            : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
            "r"(B[0]), "r"(B[1]), 
            "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
        i++;
    }
    __syncthreads();
    if (i == (block_end-1)) {
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum, alpha[2];
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<((threadIdx.x>>2)&7))-1))];
            for (int k = 0; k < 2; k++) {
                mask[k] = (float)(((col_mask>>((threadIdx.x>>2)&7))&(row_mask>>((threadIdx.x&3)*2+k)))&1);
                alpha[k] = min(-1.f+2*mask[k], sparse_A[i&1][((threadIdx.x>>3)&3)+k*8][threadIdx.x&7]+sparse_A[i&1][((threadIdx.x>>3)&3)+4+k*8][threadIdx.x&7])*scale;
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][((threadIdx.x>>2)&7)]);
            softmax[(i+1)&1][0][((threadIdx.x>>2)&7)] = alpha_max;
            alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
            alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
            alpha_sum = alpha[0] + alpha[1];
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1);
            }
            alpha_sum += softmax[i&1][1][((threadIdx.x>>2)&7)]*__expf(softmax[i&1][0][((threadIdx.x>>2)&7)]-alpha_max);
            softmax[(i+1)&1][1][((threadIdx.x>>2)&7)] = alpha_sum;
            float rcp = 1.f / (alpha_sum+1e-16f);
            alpha[0] *= rcp;
            alpha[1] *= rcp;
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(alpha[j]));
        }
        for (int j=0; j<2; j++) {
            float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j]-softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                        *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
            C[j] *= update;
            C[j+2] *= update;
        }
        for (int j=0; j<2; j++) {
            // if (SparseAToX_idx[i%3][(threadIdx.x&3)*2+j]<node_num) {
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2]) : "f"(dense_X[i%3][(threadIdx.x&3)*2+j][(((threadIdx.x>>2)&7)+(threadIdx.x>>5)*16+((threadIdx.x&3)*2+j)*4)&31]));
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2+1]) : "f"(dense_X[i%3][(threadIdx.x&3)*2+j][(((threadIdx.x>>2)&7)+(threadIdx.x>>5)*16+8+((threadIdx.x&3)*2+j)*4)&31]));
            // } else {
            //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2]) : "f"(0.f));
            //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j*2+1]) : "f"(0.f));
            // }
        }
        asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
            : "=f"(C[0]), "=f"(C[1]), "=f"(C[2]), "=f"(C[3])
            : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
            "r"(B[0]), "r"(B[1]), 
            "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
    }
    for (int j=0; j<4; j++)
        if (bid*8+(threadIdx.x&3)*2+(j&1) < node_num)
            output[(bid*8+(threadIdx.x&3)*2+(j&1))*32+((threadIdx.x&32)>>1)+((threadIdx.x>>2)&7)+(j&2)*4] = C[j];
}

__global__ void agnn_kernel_8x8_32(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[2][8];
    __shared__ float dense_X[2][8][32];
    __shared__ float softmax[2][2][8];

    uint32_t D[8], A[2], B[4], E[1] = {0xeeee4444};
    float C[8] = {0.f};
    for (int i=0; i<8; i++) {
        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(D[i]) : "f"(
            x[min(bid*8+(threadIdx.x>>2), node_num-1)*32+i*4+(threadIdx.x&3)]/x_norm[min(bid*8+(threadIdx.x>>2), node_num-1)]));
    }
    int i = block_start;
    int cur_addr;
    for (int j = 0; j < 2; j++) {
        SparseAToX_idx[i&1][(threadIdx.x>>3)+j*4] = SparseAToX[i*8+(threadIdx.x>>3)+j*4];
        cur_addr = __cvta_generic_to_shared(&dense_X[i&1][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i&1][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    softmax[i&1][0][threadIdx.x>>2] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>2] = 0.f;
    __syncthreads();
    for (; i<(block_end-1); i++) {
        for (int j = 0; j < 2; j++) {
            SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)+j*4] = SparseAToX[(i+1)*8+(threadIdx.x>>3)+j*4];
            cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)&1][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)&1][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // SDDMM
        float alpha[4] = {0.f};
        for (int k = 0; k < 2; k++) {
            for (int l = 0; l < 4; l++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                    dense_X[i&1][(threadIdx.x>>2)][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+l*2+k*16+(threadIdx.x>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][(threadIdx.x>>2)], node_num-1)]));
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum;
            int row_id = threadIdx.x>>2;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 2; k++) {
                int col_id = (threadIdx.x&3)*2+k;
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[k] = min(-1.f+2*mask[k], alpha[k]+alpha[k+2])*beta[0];
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][threadIdx.x>>2]);
            softmax[(i+1)&1][0][threadIdx.x>>2] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 2; k++) {
                alpha[k] = mask[k] * __expf(alpha[k]-alpha_max);
                alpha_sum += alpha[k];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>2] * __expf(softmax[i&1][0][threadIdx.x>>2] - alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>2] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 2; k++) {
                alpha[k] *= rcp;
            }
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j] - softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                        *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int l = 0; l < 4; l++) {
                    // if (SparseAToX_idx[i&1][(threadIdx.x&3)*2+(l>>1)] < node_num)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][(threadIdx.x&3)*2+(l>>1)][((threadIdx.x>>2)+(l>>1)*4+((l&1)+(threadIdx.x&3))*8+j*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                    : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "r"(A[0]), "r"(A[1]), 
                    "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-1)) {
        // SDDMM
        float alpha[4] = {0.f};
        for (int k = 0; k < 2; k++) {
            for (int l = 0; l < 4; l++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(
                    dense_X[i&1][(threadIdx.x>>2)][((threadIdx.x&1)*8+((threadIdx.x&2)>>1)+l*2+k*16+(threadIdx.x>>2)*4)&31]/x_norm[min(SparseAToX_idx[i&1][(threadIdx.x>>2)], node_num-1)]));
            asm volatile("mma.sp::ordered_metadata.sync.aligned.m16n8k16.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, "
                "{%8, %9, %10, %11}, {%12, %13, %14, %15}, %16, 0x0;\n"
                : "=f"(alpha[0]), "=f"(alpha[1]), "=f"(alpha[2]), "=f"(alpha[3])
                : "r"(D[k*4]), "r"(D[k*4+2]), "r"(D[k*4+1]), "r"(D[k*4+3]), 
                "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                "f"(alpha[0]), "f"(alpha[1]), "f"(alpha[2]), "f"(alpha[3]), "r"(E[0]));
        }
        // Softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i];
            float mask[2], alpha_max, alpha_sum;
            int row_id = threadIdx.x>>2;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            for (int k = 0; k < 2; k++) {
                int col_id = (threadIdx.x&3)*2+k;
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                alpha[k] = min(-1.f+2*mask[k], alpha[k]+alpha[k+2])*beta[0];
            }
            alpha_max = max(alpha[0], alpha[1]);
            for (int k = 1; k < 4; k<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*k-1, 4));
            }
            alpha_max = max(alpha_max, softmax[i&1][0][threadIdx.x>>2]);
            softmax[(i+1)&1][0][threadIdx.x>>2] = alpha_max;
            alpha_sum = 0;
            for (int k = 0; k < 2; k++) {
                alpha[k] = mask[k] * __expf(alpha[k]-alpha_max);
                alpha_sum += alpha[k];
            }
            for (int k = 1; k < 4; k<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*k-1, 4);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>2] * __expf(softmax[i&1][0][threadIdx.x>>2] - alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>2] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            for (int k = 0; k < 2; k++) {
                alpha[k] *= rcp;
            }
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(threadIdx.x&3)*2+j] - softmax[(i+1)&1][0][(threadIdx.x&3)*2+j])
                        *(softmax[i&1][1][(threadIdx.x&3)*2+j]+1e-16f)/(softmax[(i+1)&1][1][(threadIdx.x&3)*2+j]+1e-16f);
                for (int k = 0; k < 4; k++) 
                    C[j+k*2] *= update;
            }
            for (int j = 0; j < 2; j++) {
                for (int l = 0; l < 4; l++) {
                    // if (SparseAToX_idx[i&1][(threadIdx.x&3)*2+(l>>1)] < node_num)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(dense_X[i&1][(threadIdx.x&3)*2+(l>>1)][((threadIdx.x>>2)+(l>>1)*4+((l&1)+(threadIdx.x&3))*8+j*16)&31]));
                    // else
                    //     asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l]) : "f"(0.f));
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(C[j*4]), "=f"(C[j*4+1]), "=f"(C[j*4+2]), "=f"(C[j*4+3])
                    : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]), 
                    "r"(A[0]), "r"(A[1]), 
                    "f"(C[j*4]), "f"(C[j*4+1]), "f"(C[j*4+2]), "f"(C[j*4+3]));
            }
        }
    }
    for (int j = 0; j < 8; j++)
        if (bid*8+(threadIdx.x&3)*2+(j&1)<node_num)
            output[(bid*8+(threadIdx.x&3)*2+(j&1))*32+(j>>1)*8+(threadIdx.x>>2)] = C[j];
}

__global__ void agnn_kernel_4x8_32_3(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[3][8];
    __shared__ float dense_X[3][8][32];
    __shared__ float softmax[2][2][4];
    __shared__ float sparse_A[2][4][8];

    float D[4], C[4] = {0.f, 0.f, 0.f, 0.f};
    for (int j=0; j<4; j++) {
        D[j] = x[min(bid*4+(threadIdx.x>>3), node_num-1)*32+(threadIdx.x&7)*4+j]/x_norm[min(bid*4+(threadIdx.x>>3), node_num-1)];
    }
    int i = block_start;
    for (int j=0; j<2; j++) {
        SparseAToX_idx[i%3][(threadIdx.x>>3)+j*4] = SparseAToX[i*8+(threadIdx.x>>3)+j*4];
        int cur_addr = __cvta_generic_to_shared(&dense_X[i%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    if ((i+1)<block_end) {
        for (int j=0; j<2; j++) {
            SparseAToX_idx[(i+1)%3][(threadIdx.x>>3)+j*4] = SparseAToX[(i+1)*8+(threadIdx.x>>3)+j*4];
            int cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
            asm volatile("cp.async.commit_group;\n"::);
        }
        float alpha = 0.f, norm = 1.f/x_norm[min(SparseAToX_idx[i%3][threadIdx.x&7], node_num-1)];
        for (int j=0; j<32; j++) {
            alpha += __shfl_sync(FULL_MASK, D[j&3], (j>>2), 8) * dense_X[i%3][threadIdx.x&7][((threadIdx.x&7)*4+j)&31];
        }
        sparse_A[i&1][threadIdx.x>>3][threadIdx.x&7] = alpha * norm;
    }
    softmax[i&1][0][threadIdx.x>>3] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>3] = 0.f;
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    for (; i<(block_end-2); i++) {
        for (int j=0; j<2; j++) {
            SparseAToX_idx[(i+2)%3][(threadIdx.x>>3)+j*4] = SparseAToX[(i+2)*8+(threadIdx.x>>3)+j*4];
            int cur_addr = __cvta_generic_to_shared(&dense_X[(i+2)%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+2)%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
            asm volatile("cp.async.commit_group;\n"::);
        }
        {
            float alpha = 0.f, norm = 1.f/x_norm[min(SparseAToX_idx[(i+1)%3][threadIdx.x&7], node_num-1)];
            for (int j=0; j<32; j++) {
                alpha += __shfl_sync(FULL_MASK, D[j&3], (j>>2), 8) * dense_X[(i+1)%3][threadIdx.x&7][((threadIdx.x&7)*4+j)&31];
            }
            sparse_A[(i+1)&1][threadIdx.x>>3][threadIdx.x&7] = alpha * norm;
        }      
        {
            int row_start = BitMaskRowOffset[i];
            uint32_t col_mask = BitColMask[i/2]>>(4*(i&1));
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<(threadIdx.x>>3))-1))];
            float mask = (col_mask>>(threadIdx.x>>3))&(row_mask>>(threadIdx.x&7))&1;
            float alpha = min(-1.f+2*mask, sparse_A[i&1][threadIdx.x>>3][threadIdx.x&7])*beta[0];
            float alpha_max = max(alpha, softmax[i&1][0][threadIdx.x>>3]);
            for (int j=1;  j<8; j<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
            }
            softmax[(i+1)&1][0][threadIdx.x>>3] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1;  j<8; j<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>3]*__expf(softmax[i&1][0][threadIdx.x>>3]-alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>3] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            alpha *= rcp;
            float update = __expf(softmax[i&1][0][threadIdx.x>>3]-alpha_max) *
                        (softmax[i&1][1][threadIdx.x>>3]+1e-16f)*rcp;
            for (int j=0; j<4; j++) {
                C[j] *= update; 
                for (int k=0; k<8; k++)
                    C[j] += __shfl_sync(FULL_MASK, alpha, k, 8) * dense_X[i%3][k][(j+((threadIdx.x&7)+k)*4)&31];
            }    
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-2)) {
        {
            float alpha = 0.f, norm = 1.f/x_norm[min(SparseAToX_idx[(i+1)%3][threadIdx.x&7], node_num-1)];
            for (int j=0; j<32; j++) {
                alpha += __shfl_sync(FULL_MASK, D[j&3], (j>>2), 8) * dense_X[(i+1)%3][threadIdx.x&7][((threadIdx.x&7)*4+j)&31];
            }
            sparse_A[(i+1)&1][threadIdx.x>>3][threadIdx.x&7] = alpha * norm;
        }
        {
            int row_start = BitMaskRowOffset[i];
            uint32_t col_mask = BitColMask[i/2]>>(4*(i&1));
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<(threadIdx.x>>3))-1))];
            float mask = (col_mask>>(threadIdx.x>>3))&(row_mask>>(threadIdx.x&7))&1;
            float alpha = min(-1.f+2*mask, sparse_A[i&1][threadIdx.x>>3][threadIdx.x&7])*beta[0];
            float alpha_max = max(alpha, softmax[i&1][0][threadIdx.x>>3]);
            for (int j=1;  j<8; j<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
            }
            softmax[(i+1)&1][0][threadIdx.x>>3] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1;  j<8; j<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>3]*__expf(softmax[i&1][0][threadIdx.x>>3]-alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>3] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            alpha *= rcp;
            float update = __expf(softmax[i&1][0][threadIdx.x>>3]-alpha_max) *
                        (softmax[i&1][1][threadIdx.x>>3]+1e-16f)*rcp;
            for (int j=0; j<4; j++) {
                C[j] *= update; 
                for (int k=0; k<8; k++)
                    C[j] += __shfl_sync(FULL_MASK, alpha, k, 8) * dense_X[i%3][k][(j+((threadIdx.x&7)+k)*4)&31];
            }    
        }
        i++; 
    }
    __syncthreads();
    if (i == (block_end - 1)) {
        int row_start = BitMaskRowOffset[i];
        uint32_t col_mask = BitColMask[i/2]>>(4*(i&1));
        uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<(threadIdx.x>>3))-1))];
        float mask = (col_mask>>(threadIdx.x>>3))&(row_mask>>(threadIdx.x&7))&1;
        float alpha = min(-1.f+2*mask, sparse_A[i&1][threadIdx.x>>3][threadIdx.x&7])*beta[0];
        float alpha_max = max(alpha, softmax[i&1][0][threadIdx.x>>3]);
        for (int j=1;  j<8; j<<=1) {
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
        }
        softmax[(i+1)&1][0][threadIdx.x>>3] = alpha_max;
        alpha = mask * __expf(alpha-alpha_max);
        float alpha_sum = alpha;
        for (int j=1;  j<8; j<<=1) {
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
        }
        alpha_sum += softmax[i&1][1][threadIdx.x>>3]*__expf(softmax[i&1][0][threadIdx.x>>3]-alpha_max);
        softmax[(i+1)&1][1][threadIdx.x>>3] = alpha_sum;
        float rcp = 1.f/(alpha_sum+1e-16f);
        alpha *= rcp;
        float update = __expf(softmax[i&1][0][threadIdx.x>>3]-alpha_max) *
                    (softmax[i&1][1][threadIdx.x>>3]+1e-16f)*rcp;
        for (int j=0; j<4; j++) {
            C[j] *= update; 
            for (int k=0; k<8; k++)
                C[j] += __shfl_sync(FULL_MASK, alpha, k, 8) * dense_X[i%3][k][(j+((threadIdx.x&7)+k)*4)&31];
        } 
    }
    if (bid*4+(threadIdx.x>>3)<node_num)
        FLOAT4(output[(bid*4+(threadIdx.x>>3))*32+(threadIdx.x&7)*4]) = FLOAT4(C[0]);
}

__global__ void agnn_kernel_2x16_32_3(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint16_t* __restrict__ BitRowMask,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];
    if(block_start == block_end) return;

    __shared__ int SparseAToX_idx[3][16];
    __shared__ float dense_X[3][16][32];
    __shared__ float softmax[2][2][2];
    __shared__ float sparse_A[2][2][16];

    float D[2], C[2] = {0.f, 0.f};
    for (int j=0; j<2; j++) {
        D[j] = x[min(bid*2+(threadIdx.x>>4), node_num-1)*32+(threadIdx.x&15)*2+j]/x_norm[min(bid*2+(threadIdx.x>>4), node_num-1)];
    }
    int i = block_start;
    for (int j=0; j<4; j++) {
        SparseAToX_idx[i%3][(threadIdx.x>>3)+j*4] = SparseAToX[i*16+(threadIdx.x>>3)+j*4];
        int cur_addr = __cvta_generic_to_shared(&dense_X[i%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[i%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    if ((i+1)<block_end) {
        for (int j=0; j<4; j++) {
            SparseAToX_idx[(i+1)%3][(threadIdx.x>>3)+j*4] = SparseAToX[(i+1)*16+(threadIdx.x>>3)+j*4];
            int cur_addr = __cvta_generic_to_shared(&dense_X[(i+1)%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+1)%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
            asm volatile("cp.async.commit_group;\n"::);
        }
        float alpha = 0.f, norm = 1.f/x_norm[min(SparseAToX_idx[i%3][threadIdx.x&15], node_num-1)];
        for (int j=0; j<32; j++) {
            alpha += __shfl_sync(FULL_MASK, D[j&1], (j>>1), 16) * dense_X[i%3][threadIdx.x&15][((threadIdx.x&15)*4+j)&31];
        }
        sparse_A[i&1][threadIdx.x>>4][threadIdx.x&15] = alpha;
    }
    softmax[i&1][0][threadIdx.x>>4] = -1.0f * beta[0];
    softmax[i&1][1][threadIdx.x>>4] = 0.f;
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    for (; i<(block_end-2); i++) {
        for (int j=0; j<4; j++) {
            SparseAToX_idx[(i+2)%3][(threadIdx.x>>3)+j*4] = SparseAToX[(i+2)*16+(threadIdx.x>>3)+j*4];
            int cur_addr = __cvta_generic_to_shared(&dense_X[(i+2)%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(SparseAToX_idx[(i+2)%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
            asm volatile("cp.async.commit_group;\n"::);
        }
        {
            float alpha = 0, norm = 1.f/x_norm[min(SparseAToX_idx[(i+1)%3][threadIdx.x&15], node_num-1)];
            for (int j=0; j<32; j++) {
                alpha += __shfl_sync(FULL_MASK, D[j&1], (j>>1), 16) * dense_X[(i+1)%3][threadIdx.x&15][((threadIdx.x&15)*4+j)&31];
            }
            sparse_A[(i+1)&1][threadIdx.x>>4][threadIdx.x&15] = alpha;
        }
        {
            int row_start = BitMaskRowOffset[i];
            uint32_t col_mask = BitColMask[i/4]>>(2*(i&3));
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<(threadIdx.x>>4))-1))];
            float mask = (col_mask>>(threadIdx.x>>4))&(row_mask>>(threadIdx.x&15))&1;
            float alpha = min(-1.f+2*mask, sparse_A[i&1][threadIdx.x>>4][threadIdx.x&15])*beta[0];
            float alpha_max = max(alpha, softmax[i&1][0][threadIdx.x>>4]);
            for (int j=1;  j<16; j<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 16));
            }
            softmax[(i+1)&1][0][threadIdx.x>>4] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1;  j<16; j<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 16);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>4]*__expf(softmax[i&1][0][threadIdx.x>>4]-alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>4] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            alpha *= rcp;
            float update = __expf(softmax[i&1][0][threadIdx.x>>4]-alpha_max) *
                        (softmax[i&1][1][threadIdx.x>>4]+1e-16f)*rcp;
            for (int j=0; j<2; j++) {
                C[j] *= update; 
                for (int k=0; k<16; k++)
                    C[j] += __shfl_sync(FULL_MASK, alpha, k, 16) * dense_X[i%3][k][(j+(threadIdx.x&15)*2+k*4)&31];
            }    
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if (i == (block_end-2)) {
        {
            float alpha = 0, norm = 1.f/x_norm[min(SparseAToX_idx[(i+1)%3][threadIdx.x&15], node_num-1)];
            for (int j=0; j<32; j++) {
                alpha += __shfl_sync(FULL_MASK, D[j&1], (j>>1), 16) * dense_X[(i+1)%3][threadIdx.x&15][((threadIdx.x&15)*4+j)&31];
            }
            sparse_A[(i+1)&1][threadIdx.x>>4][threadIdx.x&15] = alpha;
        }
        {
            int row_start = BitMaskRowOffset[i];
            uint32_t col_mask = BitColMask[i/4]>>(2*(i&3));
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<(threadIdx.x>>4))-1))];
            float mask = (col_mask>>(threadIdx.x>>4))&(row_mask>>(threadIdx.x&15))&1;
            float alpha = min(-1.f+2*mask, sparse_A[i&1][threadIdx.x>>4][threadIdx.x&15])*beta[0];
            float alpha_max = max(alpha, softmax[i&1][0][threadIdx.x>>4]);
            for (int j=1;  j<16; j<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 16));
            }
            softmax[(i+1)&1][0][threadIdx.x>>4] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1;  j<16; j<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 16);
            }
            alpha_sum += softmax[i&1][1][threadIdx.x>>4]*__expf(softmax[i&1][0][threadIdx.x>>4]-alpha_max);
            softmax[(i+1)&1][1][threadIdx.x>>4] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            alpha *= rcp;
            float update = __expf(softmax[i&1][0][threadIdx.x>>4]-alpha_max) *
                        (softmax[i&1][1][threadIdx.x>>4]+1e-16f)*rcp;
            for (int j=0; j<2; j++) {
                C[j] *= update; 
                for (int k=0; k<16; k++)
                    C[j] += __shfl_sync(FULL_MASK, alpha, k, 16) * dense_X[i%3][k][(j+(threadIdx.x&15)*2+k*4)&31];
            }    
        }
        i++;
    }
    __syncthreads();
    if (i == (block_end - 1)) {
        int row_start = BitMaskRowOffset[i];
        uint32_t col_mask = BitColMask[i/4]>>(2*(i&3));
        uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<(threadIdx.x>>4))-1))];
        float mask = (col_mask>>(threadIdx.x>>4))&(row_mask>>(threadIdx.x&15))&1;
        float alpha = min(-1.f+2*mask, sparse_A[i&1][threadIdx.x>>4][threadIdx.x&15])*beta[0];
        float alpha_max = max(alpha, softmax[i&1][0][threadIdx.x>>4]);
        for (int j=1;  j<16; j<<=1) {
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 16));
        }
        softmax[(i+1)&1][0][threadIdx.x>>4] = alpha_max;
        alpha = mask * __expf(alpha-alpha_max);
        float alpha_sum = alpha;
        for (int j=1;  j<16; j<<=1) {
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 16);
        }
        alpha_sum += softmax[i&1][1][threadIdx.x>>4]*__expf(softmax[i&1][0][threadIdx.x>>4]-alpha_max);
        softmax[(i+1)&1][1][threadIdx.x>>4] = alpha_sum;
        float rcp = 1.f/(alpha_sum+1e-16f);
        alpha *= rcp;
        float update = __expf(softmax[i&1][0][threadIdx.x>>4]-alpha_max) *
                    (softmax[i&1][1][threadIdx.x>>4]+1e-16f)*rcp;
        for (int j=0; j<2; j++) {
            C[j] *= update; 
            for (int k=0; k<16; k++)
                C[j] += __shfl_sync(FULL_MASK, alpha, k, 16) * dense_X[i%3][k][(j+(threadIdx.x&15)*2+k*4)&31];
        }    
    }
    if (bid*2+(threadIdx.x>>4)<node_num)
        FLOAT2(output[(bid*2+(threadIdx.x>>4))*32+(threadIdx.x&15)*2]) = FLOAT2(C[0]);
}

at::Tensor AGNN_short(
    at::Tensor feature,
    at::Tensor RowWindowOffsets,
    at::Tensor SparseAToX,
    at::Tensor BitMaskRowOffset,
    at::Tensor BitColMask,
    at::Tensor BitRowMask,
    at::Tensor beta,
    int out_feats,
    int block_high,
    int block_width
) {
    int num_nodes = feature.size(0);
    auto x_norm = feature.norm(2, -1).clamp_min(1e-12);
    auto output = at::empty({num_nodes, out_feats}, feature.options());

    int blocks = (num_nodes + block_high - 1) / block_high;
    int mode = block_high*100+block_width;
    switch (mode) {
        case 1608:
            agnn_kernel_16x8_32<<<blocks, 64>>>(
                RowWindowOffsets.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BitMaskRowOffset.data_ptr<int>(),
                (uint16_t*)BitColMask.data_ptr<uint8_t>(),
                BitRowMask.data_ptr<uint8_t>(),
                beta.data_ptr<float>(),
                feature.data_ptr<float>(),
                x_norm.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 1616:
            agnn_kernel_16x16_32<<<blocks, 64>>>(
                RowWindowOffsets.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BitMaskRowOffset.data_ptr<int>(),
                (uint16_t*)BitColMask.data_ptr<uint8_t>(),
                (uint16_t*)BitRowMask.data_ptr<uint8_t>(),
                beta.data_ptr<float>(),
                feature.data_ptr<float>(),
                x_norm.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 816:
            agnn_kernel_8x16_32<<<blocks, 32>>>(
                RowWindowOffsets.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BitMaskRowOffset.data_ptr<int>(),
                BitColMask.data_ptr<uint8_t>(),
                (uint16_t*)BitRowMask.data_ptr<uint8_t>(),
                beta.data_ptr<float>(),
                feature.data_ptr<float>(),
                x_norm.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 808:
            agnn_kernel_8x8_32_3<<<blocks, 64>>>(
                RowWindowOffsets.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BitMaskRowOffset.data_ptr<int>(),
                BitColMask.data_ptr<uint8_t>(),
                BitRowMask.data_ptr<uint8_t>(),
                beta.data_ptr<float>(),
                feature.data_ptr<float>(),
                x_norm.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 408:
            agnn_kernel_4x8_32_3<<<blocks, 32>>>(
                RowWindowOffsets.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BitMaskRowOffset.data_ptr<int>(),
                BitColMask.data_ptr<uint8_t>(),
                BitRowMask.data_ptr<uint8_t>(),
                beta.data_ptr<float>(),
                feature.data_ptr<float>(),
                x_norm.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 216:
            agnn_kernel_2x16_32_3<<<blocks, 32>>>(
                RowWindowOffsets.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BitMaskRowOffset.data_ptr<int>(),
                BitColMask.data_ptr<uint8_t>(),
                (uint16_t*)BitRowMask.data_ptr<uint8_t>(),
                beta.data_ptr<float>(),
                feature.data_ptr<float>(),
                x_norm.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        default:
            printf("Unsupported mode: %d\n", mode);
            exit(1);
    }
    return output;
}

__global__ void agnn_csr(
    const int* __restrict__ row_offset,
    const int* __restrict__ index,
    const float* __restrict__ beta,
    const float* __restrict__ x,
    const float* __restrict__ x_norm,
    float* __restrict__ output,
    int node_num
) {
    int bid = blockIdx.x;
    int row_start = row_offset[bid];
    int row_end = row_offset[bid+1];

    __shared__ int node_index[3][8];
    __shared__ float dense_X[3][8][32];
    __shared__ float softmax[2][2];
    __shared__ float sparse_A[2][8];

    float D, C = 0.f;
    D = x[bid*32+threadIdx.x]/x_norm[bid];
    for (int j=0; j<2; j++) {
        node_index[0][(threadIdx.x>>3)+j*4] = index[min(row_start+(threadIdx.x>>3)+j*4, row_end-1)];
        int cur_addr = __cvta_generic_to_shared(&dense_X[0][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(node_index[0][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
        asm volatile("cp.async.commit_group;\n"::);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    if (8<row_end) {
        for (int j=0; j<2; j++) {
            node_index[1][(threadIdx.x>>3)+j*4] = index[min(row_start+8+(threadIdx.x>>3)+j*4, row_end-1)];
            int cur_addr = __cvta_generic_to_shared(&dense_X[1][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(node_index[1][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
            asm volatile("cp.async.commit_group;\n"::);
        }
        float alpha=0.f, norm = 1.f/x_norm[node_index[0][threadIdx.x>>2]];
        for (int j=0; j<8; j++) {
            alpha += __shfl_sync(FULL_MASK, D, (threadIdx.x&3)*8+j)*dense_X[0][(threadIdx.x>>2)][((threadIdx.x>>2)*4+(threadIdx.x&3)*8+j)&31];
        }
        for (int j=1; j<4; j<<=1) {
            alpha += __shfl_xor_sync(FULL_MASK, alpha, 2*j-1, 4);
        }
        sparse_A[0][threadIdx.x>>2] = alpha*norm;
    }
    softmax[0][0] = -1.0f * beta[0];
    softmax[0][1] = 0.f;
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    for (int i=0; i<((row_end-row_start+7)/8-2); i++) {
        for (int j=0; j<2; j++) {
            node_index[(i+2)%3][(threadIdx.x>>3)+j*4] = index[min(row_start+8*(i+2)+(threadIdx.x>>3)+j*4, row_end-1)];
            int cur_addr = __cvta_generic_to_shared(&dense_X[(i+2)%3][(threadIdx.x>>3)+j*4][(((threadIdx.x&7)+(threadIdx.x>>3)+j*4)*4)&31]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n"::"r"(cur_addr), "l"(&x[min(node_index[(i+2)%3][(threadIdx.x>>3)+j*4], node_num-1)*32+(threadIdx.x&7)*4]));
            asm volatile("cp.async.commit_group;\n"::);
        }
        float alpha=0.f, norm = 1.f/x_norm[node_index[(i+1)%3][threadIdx.x>>2]];
        for (int j=0; j<8; j++) {
            alpha += __shfl_sync(FULL_MASK, D, (threadIdx.x&3)*8+j)*dense_X[(i+1)%3][(threadIdx.x>>2)][((threadIdx.x>>2)*4+(threadIdx.x&3)*8+j)&31];
        }
        for (int j=1; j<4; j<<=1) {
            alpha += __shfl_xor_sync(FULL_MASK, alpha, 2*j-1, 4);
        }
        sparse_A[(i+1)%3][threadIdx.x>>2] = alpha*norm;
        {
            int col_id = threadIdx.x&7;
            float mask = row_start+i*8+col_id<row_end;
            float alpha = min(-1.f+2*mask, sparse_A[i&1][col_id])*beta[0];
            float alpha_max = max(alpha, softmax[i&1][0]);
            for (int j=1;  j<8; j<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 16));
            }
            softmax[(i+1)&1][0] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1;  j<8; j<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 16);
            }
            alpha_sum += softmax[i&1][1]*__expf(softmax[i&1][0]-alpha_max);
            softmax[(i+1)&1][1] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            alpha *= rcp;
            float update = __expf(softmax[i&1][0]-alpha_max) *(softmax[i&1][1]+1e-16f)*rcp;
            C *= update;
            for (int k=0; k<8; k++)
                C += __shfl_sync(FULL_MASK, alpha, k, 8) * dense_X[i%3][k][(threadIdx.x+k*4)&31];
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    if ((row_end-row_start+7)/8>=2) {
        int i = ((row_end-row_start+7)/8-2);
        float alpha=0.f, norm = 1.f/x_norm[node_index[(i+1)%3][threadIdx.x>>2]];
        for (int j=0; j<8; j++) {
            alpha += __shfl_sync(FULL_MASK, D, (threadIdx.x&3)*8+j)*dense_X[(i+1)%3][(threadIdx.x>>2)][((threadIdx.x>>2)*4+(threadIdx.x&3)*8+j)&31];
        }
        for (int j=1; j<4; j<<=1) {
            alpha += __shfl_xor_sync(FULL_MASK, alpha, 2*j-1, 4);
        }
        sparse_A[(i+1)%3][threadIdx.x>>2] = alpha*norm;
        {
            int col_id = threadIdx.x&7;
            float mask = row_start+i*8+col_id<row_end;
            float alpha = min(-1.f+2*mask, sparse_A[i&1][col_id])*beta[0];
            float alpha_max = max(alpha, softmax[i&1][0]);
            for (int j=1;  j<8; j<<=1) {
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 16));
            }
            softmax[(i+1)&1][0] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1;  j<8; j<<=1) {
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 16);
            }
            alpha_sum += softmax[i&1][1]*__expf(softmax[i&1][0]-alpha_max);
            softmax[(i+1)&1][1] = alpha_sum;
            float rcp = 1.f/(alpha_sum+1e-16f);
            alpha *= rcp;
            float update = __expf(softmax[i&1][0]-alpha_max) *(softmax[i&1][1]+1e-16f)*rcp;
            C *= update;
            for (int k=0; k<8; k++)
                C += __shfl_sync(FULL_MASK, alpha, k, 8) * dense_X[i%3][k][(threadIdx.x+k*4)&31];
        }
        __syncthreads();
    }
    if ((row_end-row_start+7)/8>=1) {
        int i = ((row_end-row_start+7)/8-1);
        int col_id = threadIdx.x&7;
        float mask = row_start+i*8+col_id<row_end;
        float alpha = min(-1.f+2*mask, sparse_A[i&1][col_id])*beta[0];
        float alpha_max = max(alpha, softmax[i&1][0]);
        for (int j=1;  j<8; j<<=1) {
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 16));
        }
        softmax[(i+1)&1][0] = alpha_max;
        alpha = mask * __expf(alpha-alpha_max);
        float alpha_sum = alpha;
        for (int j=1;  j<8; j<<=1) {
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 16);
        }
        alpha_sum += softmax[i&1][1]*__expf(softmax[i&1][0]-alpha_max);
        softmax[(i+1)&1][1] = alpha_sum;
        float rcp = 1.f/(alpha_sum+1e-16f);
        alpha *= rcp;
        float update = __expf(softmax[i&1][0]-alpha_max) *(softmax[i&1][1]+1e-16f)*rcp;
        C *= update;
        for (int k=0; k<8; k++)
            C += __shfl_sync(FULL_MASK, alpha, k, 8) * dense_X[i%3][k][(threadIdx.x+k*4)&31];
    }
    output[bid*32+threadIdx.x] = C;
}

at::Tensor AGNN_CSR(
    at::Tensor feature,
    at::Tensor beta,
    at::Tensor row_offset,
    at::Tensor index
) {
    int num_nodes = feature.size(0);
    auto x_norm = feature.norm(2, -1).clamp_min(1e-12);
    auto output = at::empty({num_nodes, 32}, feature.options());

    agnn_csr<<<num_nodes, 32>>>(
        row_offset.data_ptr<int>(),
        index.data_ptr<int>(),
        beta.data_ptr<float>(),
        feature.data_ptr<float>(),
        x_norm.data_ptr<float>(),
        output.data_ptr<float>(),
        num_nodes);
    return output;
}
