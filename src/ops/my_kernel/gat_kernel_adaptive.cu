#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include "gat.cuh"

#define FULL_MASK 0xffffffff

__device__ __forceinline__ float leaky_relu(float x) {
    return x - 0.99f * min(0.f, x);
}

__global__ void gat_kernel_8x8_64(
    const int* __restrict__ RowWindowId,
    const int* __restrict__ RowWindowRowOffset,
    const int* __restrict__ RowWindowBlockOffset,
    const int* __restrict__ RowWindowSparseAToXOffset,
    const int* __restrict__ RowWindowColMaskOffset,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BlockRowMaskOffset,
    const uint8_t* __restrict__ ColMask,
    const uint8_t* __restrict__ RowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = RowWindowId[blockIdx.x];
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5; //1 bit 
    int block_start = RowWindowBlockOffset[bid];
    int block_num = RowWindowBlockOffset[bid+1] - RowWindowBlockOffset[bid];
    int row_off = RowWindowRowOffset[bid];
    int sparseAToX_off = RowWindowSparseAToXOffset[bid];
    int colmask_off = RowWindowColMaskOffset[bid];

    if(block_num == 0) return;

    __shared__ int SparseAToX_idx[2][8];
    __shared__ float sparse_A[2][8][8];
    __shared__ float dense[2][8][64];
    __shared__ float softmax[3][2][8];

    softmax[0][0][lane_id&7] = std::numeric_limits<float>::lowest();
    softmax[0][1][lane_id&7] = 0.f;
    // load
    if (lane_id < 8)
        SparseAToX_idx[0][lane_id] = SparseAToX[sparseAToX_off+lane_id];
    for (int j=0; j<2; j++) {
        int cur_addr = __cvta_generic_to_shared(&dense[0][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[0][j*4+(threadIdx.x>>4)], node_num-1)*64+4*(lane_id&15)]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    // softmax
    {
        int row_start_byte = BlockRowMaskOffset[block_start];
        uint32_t col_mask = ColMask[block_start];
        int row_id = warp_id*4+(lane_id>>3);
        int col_id = lane_id&7;
        uint32_t row_mask = RowMask[row_start_byte+__popc(col_mask&((1<<row_id)-1))];
        float mask = (col_mask>>row_id)&(row_mask>>col_id)&1;
        float alpha = (mask)?leaky_relu(alphai[min(row_off+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[0][col_id], node_num-1)]):
                                std::numeric_limits<float>::lowest();
        float alpha_max = max(alpha, softmax[0][0][row_id]);
        for (int j=1; j<8; j<<=1)
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
        softmax[1][0][row_id] = alpha_max;
        alpha = mask * __expf(alpha-alpha_max);
        float alpha_sum = alpha;
        for (int j=1; j<8; j<<=1)
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
        alpha_sum += softmax[0][1][row_id]*__expf(softmax[0][0][row_id]-alpha_max);
        softmax[1][1][row_id] = alpha_sum;
        sparse_A[0][row_id][col_id] = alpha / (alpha_sum+1e-16f);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    for (int i=0; i<block_num-1; i++) {
        // load
        if (lane_id < 8)
            SparseAToX_idx[(i+1)&1][lane_id] = SparseAToX[sparseAToX_off+(1+i)*8+lane_id];
        for (int j=0; j<2; j++) {
            int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][j*4+(threadIdx.x>>4)], node_num-1)*64+4*(lane_id&15)]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // softmax
        {
            int row_start_byte = BlockRowMaskOffset[block_start+i+1];
            uint32_t col_mask = ColMask[colmask_off+i+1];
            int row_id = warp_id*4+(lane_id>>3);
            int col_id = lane_id&7;
            uint32_t row_mask = RowMask[row_start_byte+__popc(col_mask&((1<<row_id)-1))];
            float mask = (col_mask>>row_id)&(row_mask>>col_id)&1;
            float alpha = (mask)?leaky_relu(alphai[min(row_off+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[(i+1)&1][col_id], node_num-1)]):
                                    std::numeric_limits<float>::lowest();
            float alpha_max = max(alpha, softmax[(i+1)%3][0][row_id]);
            for (int j=1; j<8; j<<=1)
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
            softmax[(i+2)%3][0][row_id] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1; j<8; j<<=1)
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
            alpha_sum += softmax[(i+1)%3][1][row_id]*__expf(softmax[(i+1)%3][0][row_id]-alpha_max);
            softmax[(i+2)%3][1][row_id] = alpha_sum;
            sparse_A[(i+1)&1][row_id][col_id] = alpha / (alpha_sum+1e-16f);
        }
        // matmul
        {
            uint32_t B[8], A[2];
            for (int k=0; k<8; k++) {
                if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][(warp_id*32+(k&3)*8+((k&4)+(lane_id&3))*4+(lane_id>>2))&63]));
                else
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f)); 
            }
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+j*4])); 
            for (int k=0; k<2; k++) {
                float update = __expf(softmax[i%3][0][(lane_id&3)*2+k]-softmax[(i+1)%3][0][(lane_id&3)*2+k])
                            *(softmax[i%3][1][(lane_id&3)*2+k]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+k]+1e-16f);
                for (int j=0; j<4; j++)
                    D[k+j*2] *= update;
            }
            for (int j = 0; j < 2; j++)
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3]) 
                    : "r"(B[j*4]), "r"(B[j*4+1]), "r"(B[j*4+2]), "r"(B[j*4+3]), 
                    "r"(A[0]), "r"(A[1]),
                    "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }   
    int i = block_num - 1;
    // matmul
    {
        uint32_t B[8], A[2];
        for (int k=0; k<8; k++) {
            if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][(warp_id*32+(k&3)*8+((k&4)+(lane_id&3))*4+(lane_id>>2))&63]));
            else
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f)); 
        }
        for (int j = 0; j < 2; j++)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+j*4])); 
        for (int k=0; k<2; k++) {
            float update = __expf(softmax[i%3][0][(lane_id&3)*2+k]-softmax[(i+1)%3][0][(lane_id&3)*2+k])
                        *(softmax[i%3][1][(lane_id&3)*2+k]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+k]+1e-16f);
            for (int j=0; j<4; j++)
                D[k+j*2] *= update;
        }
        for (int j = 0; j < 2; j++)
            asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3]) 
                : "r"(B[j*4]), "r"(B[j*4+1]), "r"(B[j*4+2]), "r"(B[j*4+3]), 
                "r"(A[0]), "r"(A[1]),
                "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
    }
    for (int k=0; k<8; k++)
        if ((row_off+(lane_id&3)*2+(k&1))<node_num)
            out[(row_off+(lane_id&3)*2+(k&1))*64+warp_id*32+(k&6)*4+(lane_id>>2)] = D[k];
}

__global__ void gat_kernel_8x16_64(
    const int* __restrict__ RowWindowId,
    const int* __restrict__ RowWindowRowOffset,
    const int* __restrict__ RowWindowBlockOffset,
    const int* __restrict__ RowWindowSparseAToXOffset,
    const int* __restrict__ RowWindowColMaskOffset,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BlockRowMaskOffset,
    const uint8_t* __restrict__ ColMask,
    const uint8_t* __restrict__ RowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = RowWindowId[blockIdx.x];
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5; // 1 bit
    int block_start = RowWindowRowOffset[bid];
    int block_num = RowWindowBlockOffset[bid+1] - RowWindowBlockOffset[bid];
    int row_off = RowWindowRowOffset[bid];
    int sparseAToX_off = RowWindowSparseAToXOffset[bid];
    int colmask_off = RowWindowColMaskOffset[bid];

    if(block_num == 0) return;

    __shared__ int SparseAToX_idx[2][16];
    __shared__ float sparse_A[2][8][16];
    __shared__ float dense[2][16][64];
    __shared__ float softmax[3][2][8];

    softmax[0][0][warp_id*4+(lane_id>>3)] = std::numeric_limits<float>::lowest();
    softmax[0][1][warp_id*4+(lane_id>>3)] = 0.f;
    // load
    if (lane_id < 16)
        SparseAToX_idx[0][lane_id] = SparseAToX[sparseAToX_off+lane_id];
    for (int j = 0; j < 4; j++) {
        int cur_addr = __cvta_generic_to_shared(&dense[0][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[0][j*4+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    // softmax
    {
        int row_start_byte = BlockRowMaskOffset[block_start];
        uint32_t col_mask = ColMask[colmask_off];
        int row_id = warp_id*4+(lane_id>>3);
        uint8_t row_mask[2] = {RowMask[row_start_byte+__popc(col_mask&((1<<row_id)-1))*2],
                            RowMask[row_start_byte+__popc(col_mask&((1<<row_id)-1))*2+1]};
        float mask[2], alpha[2];
        int col_id[2];
        for (int j=0; j<2; j++) {
            col_id[j] = (lane_id&7)*2+j;
            mask[j] = (col_mask>>row_id)&(*((uint16_t*)row_mask)>>col_id[j])&1;
            alpha[j] = (mask[j])?leaky_relu(alphai[min(row_off+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[0][col_id[j]], node_num-1)]):
                                    std::numeric_limits<float>::lowest();
        }
        float alpha_max = max(max(alpha[0],alpha[1]), softmax[0][0][row_id]);
        for (int j=1; j<8; j<<=1)
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
        softmax[1][0][row_id] = alpha_max;
        alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
        alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
        float alpha_sum = alpha[0] + alpha[1];
        for (int j=1; j<8; j<<=1)
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
        alpha_sum += softmax[0][1][row_id]*__expf(softmax[0][0][row_id]-alpha_max);
        softmax[1][1][row_id] = alpha_sum;
        for (int j=0; j<2; j++)
            sparse_A[0][row_id][col_id[j]] = alpha[j] / (alpha_sum+1e-16f);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    for (int i=0; i<(block_num-1); i++) {
        // load
        if (lane_id < 16)
            SparseAToX_idx[(i+1)&1][lane_id] = SparseAToX[sparseAToX_off+(i+1)*16+lane_id];
        // __syncwarp();
        for (int j = 0; j < 4; j++) {
            int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][j*4+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // softmax
        {
            int row_start = BlockRowMaskOffset[block_start+i+1];
            uint32_t col_mask = ColMask[colmask_off+i+1];
            int row_id = warp_id*4+(lane_id>>3);
            uint8_t row_mask[2] = {RowMask[row_start+__popc(col_mask&((1<<row_id)-1))*2],
                            RowMask[row_start+__popc(col_mask&((1<<row_id)-1))*2+1]};
            float mask[2], alpha[2];
            int col_id[2];
            for (int j=0; j<2; j++) {
                col_id[j] = (lane_id&7)*2+j;
                mask[j] = (col_mask>>row_id)&(*((uint16_t*)row_mask)>>col_id[j])&1;
                alpha[j] = (mask[j])?leaky_relu(alphai[min(row_off+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[(i+1)&1][col_id[j]], node_num-1)]):
                                        std::numeric_limits<float>::lowest();
            }
            float alpha_max = max(max(alpha[0],alpha[1]), softmax[(i+1)%3][0][row_id]);
            for (int j=1; j<8; j<<=1)
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
            softmax[(i+2)%3][0][row_id] = alpha_max;
            alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
            alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
            float alpha_sum = alpha[0] + alpha[1];
            for (int j=1; j<8; j<<=1)
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
            alpha_sum += softmax[(i+1)%3][1][row_id]*__expf(softmax[(i+1)%3][0][row_id]-alpha_max);
            softmax[(i+2)%3][1][row_id] = alpha_sum;
            for (int j=0; j<2; j++)
                sparse_A[(i+1)&1][row_id][col_id[j]] = alpha[j] / (alpha_sum+1e-16f);
        }
        //matmul
        {
            for (int j=0; j<2; j++) {
                float update = __expf(softmax[i%3][0][(lane_id&3)*2+j]-softmax[(i+1)%3][0][(lane_id&3)*2+j]) *
                            (softmax[i%3][1][(lane_id&3)*2+j]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+j]+1e-16f);
                for (int k=0; k<4; k++)
                    D[j+2*k] *= update;
            }
            uint32_t B[4], A[2];
            for (int j=0; j<2; j++) {
                for (int k=0; k<2; k++) {
                    for (int l=0; l<2; l++) {
                        if (SparseAToX_idx[i&1][(lane_id&3)+(k*2+l)*4] < node_num) {
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+(lane_id&3)*4+(lane_id>>2))&63]));
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+8+(lane_id&3)*4+(lane_id>>2))&63]));
                        } else {
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(0.f));
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(0.f));
                        }
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[l]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+(k*2+l)*4]));
                    }
                    asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                        : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3])
                        : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                        "r"(A[0]), "r"(A[1]),
                        "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
                }
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    int i = block_num - 1;
    //matmul
    {
        for (int j=0; j<2; j++) {
            float update = __expf(softmax[i%3][0][(lane_id&3)*2+j]-softmax[(i+1)%3][0][(lane_id&3)*2+j]) *
                        (softmax[i%3][1][(lane_id&3)*2+j]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+j]+1e-16f);
            for (int k=0; k<4; k++)
                D[j+2*k] *= update;
        }
        uint32_t B[4], A[2];
        for (int j=0; j<2; j++) {
            for (int k=0; k<2; k++) {
                for (int l=0; l<2; l++) {
                    if (SparseAToX_idx[i&1][(lane_id&3)+(k*2+l)*4] < node_num) {
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+(lane_id&3)*4+(lane_id>>2))&63]));
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+8+(lane_id&3)*4+(lane_id>>2))&63]));
                    } else {
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(0.f));
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(0.f));
                    }
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[l]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+(k*2+l)*4]));
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3])
                    : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                    "r"(A[0]), "r"(A[1]),
                    "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
            }
        }
    }
    for (int k=0; k<8; k++)
        if ((row_off+(lane_id&3)*2+(k&1))<node_num)
            out[(row_off+(lane_id&3)*2+(k&1))*64+warp_id*32+(k&6)*4+(lane_id>>2)] = D[k];
}

__global__ void gat_kernel_16x8_64(
    const int* __restrict__ RowWindowId,
    const int* __restrict__ RowWindowRowOffset,
    const int* __restrict__ RowWindowBlockOffset,
    const int* __restrict__ RowWindowSparseAToXOffset,
    const int* __restrict__ RowWindowColMaskOffset,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BlockRowMaskOffset,
    const uint8_t* __restrict__ ColMask,
    const uint8_t* __restrict__ RowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = RowWindowId[blockIdx.x];
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowBlockOffset[bid];
    int block_num = RowWindowBlockOffset[bid+1]-RowWindowBlockOffset[bid];
    int sparseAToX_off = RowWindowSparseAToXOffset[bid];
    int row_off = RowWindowRowOffset[bid];
    int colmask_off = RowWindowColMaskOffset[bid];

    if(block_num == 0) return;

    __shared__ int SparseAToX_idx[2][8];
    __shared__ float dense[2][8][64];
    __shared__ float softmax[2][2][16];

    SparseAToX_idx[0][threadIdx.x>>4] = SparseAToX[sparseAToX_off + (threadIdx.x>>4)];

    softmax[0][0][lane_id&15] = std::numeric_limits<float>::lowest();
    softmax[0][1][lane_id&15] = 0.f;
    int cur_addr = __cvta_generic_to_shared(&dense[0][threadIdx.x>>4][(lane_id&15)*4]);
    asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[0][threadIdx.x>>4], node_num-1)*64+4*(lane_id&15)]));
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    uint32_t B[4];
    uint32_t A[4];

    for (int i=0; i<(block_num-1); i++) {
        // load
        SparseAToX_idx[(i+1)&1][threadIdx.x>>4] = SparseAToX[sparseAToX_off+(i+1)*8+(threadIdx.x>>4)];
        int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][threadIdx.x>>4][(lane_id&15)*4]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][(threadIdx.x>>4)], node_num-1)*64+4*(lane_id&15)]));
        asm volatile("cp.async.commit_group;\n"::);
        for (int k = 0; k < 4; k++) {
            if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][warp_id*16+(k&1)*8+(lane_id>>2)]));
            else
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f));
        }
        // softmax
        {
            int row_start = BlockRowMaskOffset[block_start+i];
            float alpha[4];
            uint8_t col_mask[2] = {ColMask[colmask_off+i*2], ColMask[colmask_off+i*2+1]};
            float mask[4], alpha_max[2], alpha_sum[2];
            for (int k = 0; k < 4; k++) {
                int row_id = (lane_id>>2)+(k&1)*8;
                int col_id = (lane_id&3)+(k&2)*2;
                uint32_t row_mask = RowMask[row_start+__popc(*((uint16_t*)col_mask)&((1<<row_id)-1))];
                mask[k] = (*((uint16_t*)col_mask)>>row_id)&(row_mask>>col_id)&1;
                if (mask[k])
                    alpha[k] = leaky_relu(alphai[row_off + row_id] + alphaj[SparseAToX_idx[i&1][col_id]]);
                else
                    alpha[k] = std::numeric_limits<float>::lowest();
            }
            for (int k = 0; k < 2; k++) {
                alpha_max[k] = max(alpha[k], alpha[2+k]);
                for (int j = 1; j < 4; j*=2) {
                    alpha_max[k] = max(alpha_max[k], __shfl_xor_sync(FULL_MASK, alpha_max[k], 2*j-1, 4));
                }
                alpha_max[k] = max(alpha_max[k], softmax[i&1][0][(lane_id>>2)+k*8]);
                softmax[(i+1)&1][0][(lane_id>>2)+k*8] = alpha_max[k];
                alpha[k] = mask[k] * __expf(alpha[k] - alpha_max[k]);
                alpha[2+k] = mask[2+k] * __expf(alpha[2+k] - alpha_max[k]);
                alpha_sum[k] = alpha[k] + alpha[2+k];
                for (int j = 1; j < 4; j*=2) {
                    alpha_sum[k] += __shfl_xor_sync(FULL_MASK, alpha_sum[k], 2*j-1, 4);
                }
                alpha_sum[k] += softmax[i&1][1][(lane_id>>2)+k*8] * __expf(softmax[i&1][0][(lane_id>>2)+k*8] - alpha_max[k]);
                softmax[(i+1)&1][1][(lane_id>>2)+k*8] = alpha_sum[k];
                float rcp = 1.f/(alpha_sum[k]+1e-16f);
                alpha[k] *= rcp;
                alpha[2+k] *= rcp;
            }
            for (int k = 0; k < 4; k++) 
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[k]) : "f"(alpha[k]));
        }
        // Matmul
        {
            for (int k = 0; k < 2; k++) {
                float update = __expf(softmax[i&1][0][(lane_id>>2)+k*8]-softmax[(i+1)&1][0][(lane_id>>2)+k*8])*(softmax[i&1][1][(lane_id>>2)+k*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+k*8]+1e-16f);
                for (int j = 0; j < 4; j++) {
                    D[k*2+(j&1)+(j&2)*2] *= update;
                }
            }
            for (int k = 0; k < 2; k++) {
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                    : "=f"(D[k*4]), "=f"(D[k*4+1]), "=f"(D[k*4+2]), "=f"(D[k*4+3]) 
                    : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
                    "r"(B[k]), "r"(B[2+k])
                    "f"(D[k*4]), "f"(D[k*4+1]), "f"(D[k*4+2]), "f"(D[k*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    int i = block_num - 1;
    for (int k = 0; k < 4; k++) {
        if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][warp_id*16+(k&1)*8+(lane_id>>2)]));
        else
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f));
    }
    // softmax
    {
        int row_start = BlockRowMaskOffset[block_start+i];
        float alpha[4];
        uint8_t col_mask[2] = {ColMask[colmask_off+i*2], ColMask[colmask_off+i*2+1]};
        float mask[4], alpha_max[2], alpha_sum[2];
        for (int k = 0; k < 4; k++) {
            int row_id = (lane_id>>2)+(k&1)*8;
            int col_id = (lane_id&3)+(k&2)*2;
            uint32_t row_mask = RowMask[row_start+__popc(*((uint16_t*)col_mask)&((1<<row_id)-1))];
            mask[k] = (*((uint16_t*)col_mask)>>row_id)&(row_mask>>col_id)&1;
            if (mask[k])
                alpha[k] = leaky_relu(alphai[row_off + row_id] + alphaj[SparseAToX_idx[i&1][col_id]]);
            else
                alpha[k] = std::numeric_limits<float>::lowest();
        }
        for (int k = 0; k < 2; k++) {
            alpha_max[k] = max(alpha[k], alpha[2+k]);
            for (int j = 1; j < 4; j*=2) {
                alpha_max[k] = max(alpha_max[k], __shfl_xor_sync(FULL_MASK, alpha_max[k], 2*j-1, 4));
            }
            alpha_max[k] = max(alpha_max[k], softmax[i&1][0][(lane_id>>2)+k*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+k*8] = alpha_max[k];
            alpha[k] = mask[k] * __expf(alpha[k] - alpha_max[k]);
            alpha[2+k] = mask[2+k] * __expf(alpha[2+k] - alpha_max[k]);
            alpha_sum[k] = alpha[k] + alpha[2+k];
            for (int j = 1; j < 4; j*=2) {
                alpha_sum[k] += __shfl_xor_sync(FULL_MASK, alpha_sum[k], 2*j-1, 4);
            }
            alpha_sum[k] += softmax[i&1][1][(lane_id>>2)+k*8] * __expf(softmax[i&1][0][(lane_id>>2)+k*8] - alpha_max[k]);
            softmax[(i+1)&1][1][(lane_id>>2)+k*8] = alpha_sum[k];
            float rcp = 1.f/(alpha_sum[k]+1e-16f);
            alpha[k] *= rcp;
            alpha[2+k] *= rcp;
        }
        for (int k = 0; k < 4; k++) 
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[k]) : "f"(alpha[k]));
    }
    // Matmul
    {
        for (int k = 0; k < 2; k++) {
            float update = __expf(softmax[i&1][0][(lane_id>>2)+k*8]-softmax[(i+1)&1][0][(lane_id>>2)+k*8])*(softmax[i&1][1][(lane_id>>2)+k*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+k*8]+1e-16f);
            for (int j = 0; j < 4; j++) {
                D[k*2+(j&1)+(j&2)*2] *= update;
            }
        }
        for (int k = 0; k < 2; k++) {
            asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                : "=f"(D[k*4]), "=f"(D[k*4+1]), "=f"(D[k*4+2]), "=f"(D[k*4+3]) 
                : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
                "r"(B[k]), "r"(B[2+k])
                "f"(D[k*4]), "f"(D[k*4+1]), "f"(D[k*4+2]), "f"(D[k*4+3]));
        }
    }
    for (int k = 0; k < 4; k++) 
        if ((row_off+(lane_id>>2)+(k&1)*8) < node_num)
            *(float2*)(&out[(row_off+(lane_id>>2)+(k&1)*8)*64+warp_id*16+(k&2)*4+(lane_id&3)*2]) = *(float2*)(&D[k*2]);
}

__global__ void gat_kernel_16x16_64(
    const int* __restrict__ RowWindowId,
    const int* __restrict__ RowWindowRowOffset,
    const int* __restrict__ RowWindowBlockOffset,
    const int* __restrict__ RowWindowSparseAToXOffset,
    const int* __restrict__ RowWindowColMaskOffset,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BlockRowMaskOffset,
    const uint8_t* __restrict__ ColMask,
    const uint8_t* __restrict__ RowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = RowWindowId[blockIdx.x];
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowBlockOffset[bid];
    int block_num = RowWindowBlockOffset[bid+1] - RowWindowBlockOffset[bid];
    int row_off = RowWindowRowOffset[bid];
    int sparseAToX_off = RowWindowSparseAToXOffset[bid];
    int colmask_off = RowWindowColMaskOffset[bid];

    if (block_num == 0) return;

    __shared__ int SparseAToX_idx[2][16];
    __shared__ float dense[2][16][64];
    __shared__ float softmax[2][2][16];

    softmax[0][0][lane_id&15] = std::numeric_limits<float>::lowest();
    softmax[0][1][lane_id&15] = 0.f;

    for (int j = 0; j < 2; j++) {
        SparseAToX_idx[0][(threadIdx.x>>4)+j*8] = SparseAToX[sparseAToX_off+j*8+(threadIdx.x>>4)];
        int cur_addr = __cvta_generic_to_shared(&dense[0][j*8+(threadIdx.x>>4)][(lane_id&15)*4]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[0][j*8+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    uint32_t A[8], B[8];
    for (int i=0; i<(block_num-1); i++) {
        // load
        for (int j = 0; j < 2; j++) {
            SparseAToX_idx[(i+1)&1][(threadIdx.x>>4)+j*8] = SparseAToX[sparseAToX_off+(i+1)*16+8*j+(threadIdx.x>>4)];
            int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][j*8+(threadIdx.x>>4)][(lane_id&15)*4]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][j*8+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        for (int j = 0; j < 8; j++) {
            if (SparseAToX_idx[i&1][(lane_id&3)+(j&6)*2] < node_num)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense[i&1][(lane_id&3)+(j&6)*2][warp_id*16+(j&1)*8+(lane_id>>2)]));
            else
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));
        }
        // softmax
        {
            int row_start_byte = BlockRowMaskOffset[i+block_start];
            float alpha[8];
            uint8_t col_mask[2] = {ColMask[colmask_off+i*2], ColMask[colmask_off+i*2+1]};
            float mask[8], alpha_max[2], alpha_sum[2];
            for (int j = 0; j < 8; j++) {
                int row_id = (lane_id>>2)+(j&1)*8;
                int col_id = (lane_id&3)+(j&6)*2;
                uint8_t row_mask[2] = {RowMask[row_start_byte+__popc(*((uint16_t*)col_mask)&((1<<row_id)-1))*2],
                                RowMask[row_start_byte+__popc(*((uint16_t*)col_mask)&((1<<row_id)-1))*2+1]};
                mask[j] = (*((uint16_t*)col_mask)>>row_id)&(*((uint16_t*)row_mask)>>col_id)&1;
                if (mask[j])
                    alpha[j] = leaky_relu(alphai[min(row_off+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[i&1][col_id], node_num-1)]);
                else
                    alpha[j] = std::numeric_limits<float>::lowest();
            }
            for (int j = 0; j < 2; j++) {
                alpha_max[j] = max(alpha[j], alpha[2+j]);
                alpha_max[j] = max(alpha_max[j], max(alpha[4+j], alpha[6+j]));
                for (int k = 1; k < 4; k*=2) {
                    alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
                }
                alpha_max[j] = max(alpha_max[j], softmax[i&1][0][(lane_id>>2)+j*8]);
                softmax[(i+1)&1][0][(lane_id>>2)+j*8] = alpha_max[j];
                alpha[j] = mask[j] * __expf(alpha[j] - alpha_max[j]);
                alpha[2+j] = mask[2+j] * __expf(alpha[2+j] - alpha_max[j]);
                alpha[4+j] = mask[4+j] * __expf(alpha[4+j] - alpha_max[j]);
                alpha[6+j] = mask[6+j] * __expf(alpha[6+j] - alpha_max[j]);
                alpha_sum[j] = alpha[j] + alpha[2+j] + alpha[4+j] + alpha[6+j];
                for (int k = 1; k < 4; k*=2) {
                    alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
                }
                alpha_sum[j] += softmax[i&1][1][(lane_id>>2)+j*8] * __expf(softmax[i&1][0][(lane_id>>2)+j*8] - alpha_max[j]);
                softmax[(i+1)&1][1][(lane_id>>2)+j*8] = alpha_sum[j];
                float rcp = 1.f/(alpha_sum[j]+1e-16f);
                alpha[j] *= rcp;
                alpha[2+j] *= rcp;
                alpha[4+j] *= rcp;
                alpha[6+j] *= rcp;
            }
            for (int j = 0; j < 8; j++) {
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
            }
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(lane_id>>2)+j*8]-softmax[(i+1)&1][0][(lane_id>>2)+j*8])*(softmax[i&1][1][(lane_id>>2)+j*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+j*8]+1e-16f);
                for (int k = 0; k < 4; k++) {
                    D[j*2+(k&1)+(k&2)*2] *= update;
                }
            }
            for (int j = 0; j < 4; j++) {
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                    : "=f"(D[(j&1)*4]), "=f"(D[(j&1)*4+1]), "=f"(D[(j&1)*4+2]), "=f"(D[(j&1)*4+3]) 
                    : "r"(A[(j&2)*2]), "r"(A[(j&2)*2+1]), "r"(A[(j&2)*2+2]), "r"(A[(j&2)*2+3]), 
                    "r"(B[(j&2)*2+(j&1)]), "r"(B[2+(j&2)*2+(j&1)]),
                    "f"(D[(j&1)*4]), "f"(D[(j&1)*4+1]), "f"(D[(j&1)*4+2]), "f"(D[(j&1)*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    int i = block_num-1;
    for (int j = 0; j < 8; j++) {
        if (SparseAToX_idx[i&1][(lane_id&3)+(j&6)*2] < node_num)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense[i&1][(lane_id&3)+(j&6)*2][warp_id*16+(j&1)*8+(lane_id>>2)]));
        else
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));
    }
    // softmax
    {
        int row_start = BlockRowMaskOffset[i+block_start];
        float alpha[8];
        uint8_t col_mask[2] = {ColMask[colmask_off+i*2], ColMask[colmask_off+i*2+1]};
        float mask[8], alpha_max[2], alpha_sum[2];
        for (int j = 0; j < 8; j++) {
            int row_id = (lane_id>>2)+(j&1)*8;
            int col_id = (lane_id&3)+(j&6)*2;
            uint8_t row_mask[2] = {RowMask[row_start+__popc(*((uint16_t*)col_mask)&((1<<row_id)-1))*2],
                                RowMask[row_start+__popc(*((uint16_t*)col_mask)&((1<<row_id)-1))*2+1]};
            mask[j] = (*((uint16_t*)col_mask)>>row_id)&(*((uint16_t*)row_mask)>>col_id)&1;
            if (mask[j])
                alpha[j] = leaky_relu(alphai[min(row_off+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[i&1][col_id], node_num-1)]);
            else
                alpha[j] = std::numeric_limits<float>::lowest();
        }
        for (int j = 0; j < 2; j++) {
            alpha_max[j] = max(alpha[j], alpha[2+j]);
            alpha_max[j] = max(alpha_max[j], max(alpha[4+j], alpha[6+j]));
            for (int k = 1; k < 4; k*=2) {
                alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
            }
            alpha_max[j] = max(alpha_max[j], softmax[i&1][0][(lane_id>>2)+j*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+j*8] = alpha_max[j];
            alpha[j] = mask[j] * __expf(alpha[j] - alpha_max[j]);
            alpha[2+j] = mask[2+j] * __expf(alpha[2+j] - alpha_max[j]);
            alpha[4+j] = mask[4+j] * __expf(alpha[4+j] - alpha_max[j]);
            alpha[6+j] = mask[6+j] * __expf(alpha[6+j] - alpha_max[j]);
            alpha_sum[j] = alpha[j] + alpha[2+j] + alpha[4+j] + alpha[6+j];
            for (int k = 1; k < 4; k*=2) {
                alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
            }
            alpha_sum[j] += softmax[i&1][1][(lane_id>>2)+j*8] * __expf(softmax[i&1][0][(lane_id>>2)+j*8] - alpha_max[j]);
            softmax[(i+1)&1][1][(lane_id>>2)+j*8] = alpha_sum[j];
            float rcp = 1.f/(alpha_sum[j]+1e-16f);
            alpha[j] *= rcp;
            alpha[2+j] *= rcp;
            alpha[4+j] *= rcp;
            alpha[6+j] *= rcp;
        }
        for (int j = 0; j < 8; j++) {
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
    }
    // Matmul
    {
        for (int j = 0; j < 2; j++) {
            float update = __expf(softmax[i&1][0][(lane_id>>2)+j*8]-softmax[(i+1)&1][0][(lane_id>>2)+j*8])*(softmax[i&1][1][(lane_id>>2)+j*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+j*8]+1e-16f);
            for (int k = 0; k < 4; k++) {
                D[j*2+(k&1)+(k&2)*2] *= update;
            }
        }
        for (int j = 0; j < 4; j++) {
            asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                : "=f"(D[(j&1)*4]), "=f"(D[(j&1)*4+1]), "=f"(D[(j&1)*4+2]), "=f"(D[(j&1)*4+3]) 
                : "r"(A[(j&2)*2]), "r"(A[(j&2)*2+1]), "r"(A[(j&2)*2+2]), "r"(A[(j&2)*2+3]), 
                "r"(B[(j&2)*2+(j&1)]), "r"(B[2+(j&2)*2+(j&1)]),
                "f"(D[(j&1)*4]), "f"(D[(j&1)*4+1]), "f"(D[(j&1)*4+2]), "f"(D[(j&1)*4+3]));
        }
    }
    for (int j = 0; j < 4; j++) 
        if ((row_off+(lane_id>>2)+(j&1)*8)<node_num)
            *(float2*)(&out[(row_off+(lane_id>>2)+(j&1)*8)*64+warp_id*16+(j&2)*4+(lane_id&3)*2]) = *(float2*)(&D[j*2]);
}

__global__ void gat_kernel_8x8_64_2_(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5; //1 bit 
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];

    if(block_end == block_start) return;

    __shared__ int SparseAToX_idx[2][8];
    __shared__ float sparse_A[2][8][8];
    __shared__ float dense[2][8][64];
    __shared__ float softmax[3][2][8];

    int i = block_start;
    softmax[i%3][0][lane_id&7] = std::numeric_limits<float>::lowest();
    softmax[i%3][1][lane_id&7] = 0.f;
    // load
    if (lane_id < 8)
        SparseAToX_idx[i&1][lane_id] = SparseAToX[i*8+lane_id];
    for (int j=0; j<2; j++) {
        int cur_addr = __cvta_generic_to_shared(&dense[i&1][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[i&1][j*4+(threadIdx.x>>4)], node_num-1)*64+4*(lane_id&15)]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    // softmax
    {
        int row_start = BitMaskRowOffset[i];
        uint32_t col_mask = BitColMask[i];
        int row_id = warp_id*4+(lane_id>>3);
        int col_id = lane_id&7;
        uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
        float mask = (col_mask>>row_id)&(row_mask>>col_id)&1;
        float alpha = (mask)?leaky_relu(alphai[min(bid*8+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[i&1][col_id], node_num-1)]):
                                std::numeric_limits<float>::lowest();
        float alpha_max = max(alpha, softmax[i%3][0][row_id]);
        for (int j=1; j<8; j<<=1)
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
        softmax[(i+1)%3][0][row_id] = alpha_max;
        alpha = mask * __expf(alpha-alpha_max);
        float alpha_sum = alpha;
        for (int j=1; j<8; j<<=1)
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
        alpha_sum += softmax[i%3][1][row_id]*__expf(softmax[i%3][0][row_id]-alpha_max);
        softmax[(i+1)%3][1][row_id] = alpha_sum;
        sparse_A[i&1][row_id][col_id] = alpha / (alpha_sum+1e-16f);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    for (; i<(block_end-1); i++) {
        // load
        if (lane_id < 8)
            SparseAToX_idx[(i+1)&1][lane_id] = SparseAToX[(i+1)*8+lane_id];
        for (int j=0; j<2; j++) {
            int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][j*4+(threadIdx.x>>4)], node_num-1)*64+4*(lane_id&15)]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // softmax
        {
            int row_start = BitMaskRowOffset[i+1];
            uint32_t col_mask = BitColMask[i+1];
            int row_id = warp_id*4+(lane_id>>3);
            int col_id = lane_id&7;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            float mask = (col_mask>>row_id)&(row_mask>>col_id)&1;
            float alpha = (mask)?leaky_relu(alphai[min(bid*8+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[(i+1)&1][col_id], node_num-1)]):
                                    std::numeric_limits<float>::lowest();
            float alpha_max = max(alpha, softmax[(i+1)%3][0][row_id]);
            for (int j=1; j<8; j<<=1)
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
            softmax[(i+2)%3][0][row_id] = alpha_max;
            alpha = mask * __expf(alpha-alpha_max);
            float alpha_sum = alpha;
            for (int j=1; j<8; j<<=1)
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
            alpha_sum += softmax[(i+1)%3][1][row_id]*__expf(softmax[(i+1)%3][0][row_id]-alpha_max);
            softmax[(i+2)%3][1][row_id] = alpha_sum;
            sparse_A[(i+1)&1][row_id][col_id] = alpha / (alpha_sum+1e-16f);
        }
        // matmul
        {
            uint32_t B[8], A[2];
            for (int k=0; k<8; k++) {
                if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][(warp_id*32+(k&3)*8+((k&4)+(lane_id&3))*4+(lane_id>>2))&63]));
                else
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f)); 
            }
            for (int j = 0; j < 2; j++)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+j*4])); 
            for (int k=0; k<2; k++) {
                float update = __expf(softmax[i%3][0][(lane_id&3)*2+k]-softmax[(i+1)%3][0][(lane_id&3)*2+k])
                            *(softmax[i%3][1][(lane_id&3)*2+k]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+k]+1e-16f);
                for (int j=0; j<4; j++)
                    D[k+j*2] *= update;
            }
            for (int j = 0; j < 2; j++)
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3]) 
                    : "r"(B[j*4]), "r"(B[j*4+1]), "r"(B[j*4+2]), "r"(B[j*4+3]), 
                    "r"(A[0]), "r"(A[1]),
                    "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    i = block_end - 1;
    // matmul
    {
        uint32_t B[8], A[2];
        for (int k=0; k<8; k++) {
            if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][(warp_id*32+(k&3)*8+((k&4)+(lane_id&3))*4+(lane_id>>2))&63]));
            else
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f)); 
        }
        for (int j = 0; j < 2; j++)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+j*4])); 
        for (int k=0; k<2; k++) {
            float update = __expf(softmax[i%3][0][(lane_id&3)*2+k]-softmax[(i+1)%3][0][(lane_id&3)*2+k])
                        *(softmax[i%3][1][(lane_id&3)*2+k]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+k]+1e-16f);
            for (int j=0; j<4; j++)
                D[k+j*2] *= update;
        }
        for (int j = 0; j < 2; j++)
            asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3]) 
                : "r"(B[j*4]), "r"(B[j*4+1]), "r"(B[j*4+2]), "r"(B[j*4+3]), 
                "r"(A[0]), "r"(A[1]),
                "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
    }
    for (int k=0; k<8; k++)
        if ((bid*8+(lane_id&3)*2+(k&1))<node_num)
            out[(bid*8+(lane_id&3)*2+(k&1))*64+warp_id*32+(k&6)*4+(lane_id>>2)] = D[k];
}

__global__ void gat_kernel_8x16_64_2_(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint8_t* __restrict__ BitColMask,
    const uint16_t* __restrict__ BitRowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5; // 1 bit
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];

    if(block_end == block_start) return;

    __shared__ int SparseAToX_idx[2][16];
    __shared__ float sparse_A[2][8][16];
    __shared__ float dense[2][16][64];
    __shared__ float softmax[3][2][8];

    int i = block_start;
    softmax[i%3][0][warp_id*4+(lane_id>>3)] = std::numeric_limits<float>::lowest();
    softmax[i%3][1][warp_id*4+(lane_id>>3)] = 0.f;
    // load
    if (lane_id < 16)
        SparseAToX_idx[i&1][lane_id] = SparseAToX[i*16+lane_id];
    for (int j = 0; j < 4; j++) {
        int cur_addr = __cvta_generic_to_shared(&dense[i&1][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[i&1][j*4+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    // softmax
    {
        int row_start = BitMaskRowOffset[i]/2;
        uint32_t col_mask = BitColMask[i];
        int row_id = warp_id*4+(lane_id>>3);
        uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
        float mask[2], alpha[2];
        int col_id[2];
        for (int j=0; j<2; j++) {
            col_id[j] = (lane_id&7)*2+j;
            mask[j] = (col_mask>>row_id)&(row_mask>>col_id[j])&1;
            alpha[j] = (mask[j])?leaky_relu(alphai[min(bid*8+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[i&1][col_id[j]], node_num-1)]):
                                    std::numeric_limits<float>::lowest();
        }
        float alpha_max = max(max(alpha[0],alpha[1]), softmax[i%3][0][row_id]);
        for (int j=1; j<8; j<<=1)
            alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
        softmax[(i+1)%3][0][row_id] = alpha_max;
        alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
        alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
        float alpha_sum = alpha[0] + alpha[1];
        for (int j=1; j<8; j<<=1)
            alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
        alpha_sum += softmax[i%3][1][row_id]*__expf(softmax[i%3][0][row_id]-alpha_max);
        softmax[(i+1)%3][1][row_id] = alpha_sum;
        for (int j=0; j<2; j++)
            sparse_A[i&1][row_id][col_id[j]] = alpha[j] / (alpha_sum+1e-16f);
    }
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    for (; i < (block_end - 1); i++) {
        // load
        if (lane_id < 16)
            SparseAToX_idx[(i+1)&1][lane_id] = SparseAToX[(i+1)*16+lane_id];
        // __syncwarp();
        for (int j = 0; j < 4; j++) {
            int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][j*4+(threadIdx.x>>4)][(((lane_id&15)+j*4+(threadIdx.x>>4))*4)&63]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][j*4+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        // softmax
        {
            int row_start = BitMaskRowOffset[i+1]/2;
            uint32_t col_mask = BitColMask[i+1];
            int row_id = warp_id*4+(lane_id>>3);
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            float mask[2], alpha[2];
            int col_id[2];
            for (int j=0; j<2; j++) {
                col_id[j] = (lane_id&7)*2+j;
                mask[j] = (col_mask>>row_id)&(row_mask>>col_id[j])&1;
                alpha[j] = (mask[j])?leaky_relu(alphai[min(bid*8+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[(i+1)&1][col_id[j]], node_num-1)]):
                                        std::numeric_limits<float>::lowest();
            }
            float alpha_max = max(max(alpha[0],alpha[1]), softmax[(i+1)%3][0][row_id]);
            for (int j=1; j<8; j<<=1)
                alpha_max = max(alpha_max, __shfl_xor_sync(FULL_MASK, alpha_max, 2*j-1, 8));
            softmax[(i+2)%3][0][row_id] = alpha_max;
            alpha[0] = mask[0] * __expf(alpha[0]-alpha_max);
            alpha[1] = mask[1] * __expf(alpha[1]-alpha_max);
            float alpha_sum = alpha[0] + alpha[1];
            for (int j=1; j<8; j<<=1)
                alpha_sum += __shfl_xor_sync(FULL_MASK, alpha_sum, 2*j-1, 8);
            alpha_sum += softmax[(i+1)%3][1][row_id]*__expf(softmax[(i+1)%3][0][row_id]-alpha_max);
            softmax[(i+2)%3][1][row_id] = alpha_sum;
            for (int j=0; j<2; j++)
                sparse_A[(i+1)&1][row_id][col_id[j]] = alpha[j] / (alpha_sum+1e-16f);
        }
        //matmul
        {
            for (int j=0; j<2; j++) {
                float update = __expf(softmax[i%3][0][(lane_id&3)*2+j]-softmax[(i+1)%3][0][(lane_id&3)*2+j]) *
                            (softmax[i%3][1][(lane_id&3)*2+j]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+j]+1e-16f);
                for (int k=0; k<4; k++)
                    D[j+2*k] *= update;
            }
            uint32_t B[4], A[2];
            for (int j=0; j<2; j++) {
                for (int k=0; k<2; k++) {
                    for (int l=0; l<2; l++) {
                        if (SparseAToX_idx[i&1][(lane_id&3)+(k*2+l)*4] < node_num) {
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+(lane_id&3)*4+(lane_id>>2))&63]));
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+8+(lane_id&3)*4+(lane_id>>2))&63]));
                        } else {
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(0.f));
                            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(0.f));
                        }
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[l]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+(k*2+l)*4]));
                    }
                    asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                        : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3])
                        : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                        "r"(A[0]), "r"(A[1]),
                        "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
                }
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    i = block_end -1;
    //matmul
    {
        for (int j=0; j<2; j++) {
            float update = __expf(softmax[i%3][0][(lane_id&3)*2+j]-softmax[(i+1)%3][0][(lane_id&3)*2+j]) *
                        (softmax[i%3][1][(lane_id&3)*2+j]+1e-16f)/(softmax[(i+1)%3][1][(lane_id&3)*2+j]+1e-16f);
            for (int k=0; k<4; k++)
                D[j+2*k] *= update;
        }
        uint32_t B[4], A[2];
        for (int j=0; j<2; j++) {
            for (int k=0; k<2; k++) {
                for (int l=0; l<2; l++) {
                    if (SparseAToX_idx[i&1][(lane_id&3)+(k*2+l)*4] < node_num) {
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+(lane_id&3)*4+(lane_id>>2))&63]));
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(dense[i&1][(lane_id&3)+(k*2+l)*4][(warp_id*32+(k*2+l+j)*16+8+(lane_id&3)*4+(lane_id>>2))&63]));
                    } else {
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2]) : "f"(0.f));
                        asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[l*2+1]) : "f"(0.f));
                    }
                    asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[l]) : "f"(sparse_A[i&1][lane_id>>2][(lane_id&3)+(k*2+l)*4]));
                }
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n"
                    : "=f"(D[j*4]), "=f"(D[j*4+1]), "=f"(D[j*4+2]), "=f"(D[j*4+3])
                    : "r"(B[0]), "r"(B[1]), "r"(B[2]), "r"(B[3]),
                    "r"(A[0]), "r"(A[1]),
                    "f"(D[j*4]), "f"(D[j*4+1]), "f"(D[j*4+2]), "f"(D[j*4+3]));
            }
        }
    }
    for (int k=0; k<8; k++)
        if ((bid*8+(lane_id&3)*2+(k&1))<node_num)
            out[(bid*8+(lane_id&3)*2+(k&1))*64+warp_id*32+(k&6)*4+(lane_id>>2)] = D[k];
}

__global__ void gat_kernel_16x8_64_2_(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint16_t* __restrict__ BitColMask,
    const uint8_t* __restrict__ BitRowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];

    if(block_end == block_start) return;

    int lane_0_4 = lane_id&15;
    int thread_4_7 = threadIdx.x>>4;

    __shared__ int SparseAToX_idx[2][8];
    __shared__ float dense[2][8][64];
    __shared__ float softmax[2][2][16];
    
    int i = block_start;
    SparseAToX_idx[i&1][thread_4_7] = SparseAToX[i * 8 + thread_4_7];
    // if (warp_id == 0) {
    softmax[i&1][0][lane_0_4] = std::numeric_limits<float>::lowest();
    softmax[i&1][1][lane_0_4] = 0.f;
    // }
    int cur_addr = __cvta_generic_to_shared(&dense[i&1][thread_4_7][lane_0_4*4]);
    asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[i&1][thread_4_7], node_num-1)*64+4*lane_0_4]));
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    uint32_t B[4];
    uint32_t A[4];

    for (i = block_start; i < (block_end - 1); i++) {
        // load and init
        SparseAToX_idx[(i+1)&1][thread_4_7] = SparseAToX[(i+1)*8 + thread_4_7];
        int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][thread_4_7][lane_0_4*4]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][thread_4_7], node_num-1)*64+4*lane_0_4]));
        asm volatile("cp.async.commit_group;\n"::);
        for (int k = 0; k < 4; k++) {
            if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][warp_id*16+(k&1)*8+(lane_id>>2)]));
            else
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f));
        }
        // softmax
        {
            int row_start = BitMaskRowOffset[i];
            int row_end = BitMaskRowOffset[i+1];
            float alpha[4];
            uint32_t col_mask = BitColMask[i];
            float mask[4], alpha_max[2], alpha_sum[2];
            for (int k = 0; k < 4; k++) {
                int row_id = (lane_id>>2)+(k&1)*8;
                int col_id = (lane_id&3)+(k&2)*2;
                uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
                mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                if (mask[k])
                    alpha[k] = leaky_relu(alphai[bid * 16 + row_id] + alphaj[SparseAToX_idx[i&1][col_id]]);
                else
                    alpha[k] = std::numeric_limits<float>::lowest();
            }
            for (int k = 0; k < 2; k++) {
                alpha_max[k] = max(alpha[k], alpha[2+k]);
                for (int j = 1; j < 4; j*=2) {
                    alpha_max[k] = max(alpha_max[k], __shfl_xor_sync(FULL_MASK, alpha_max[k], 2*j-1, 4));
                }
                alpha_max[k] = max(alpha_max[k], softmax[i&1][0][(lane_id>>2)+k*8]);
                softmax[(i+1)&1][0][(lane_id>>2)+k*8] = alpha_max[k];
                alpha[k] = mask[k] * __expf(alpha[k] - alpha_max[k]);
                alpha[2+k] = mask[2+k] * __expf(alpha[2+k] - alpha_max[k]);
                alpha_sum[k] = alpha[k] + alpha[2+k];
                for (int j = 1; j < 4; j*=2) {
                    alpha_sum[k] += __shfl_xor_sync(FULL_MASK, alpha_sum[k], 2*j-1, 4);
                }
                alpha_sum[k] += softmax[i&1][1][(lane_id>>2)+k*8] * __expf(softmax[i&1][0][(lane_id>>2)+k*8] - alpha_max[k]);
                softmax[(i+1)&1][1][(lane_id>>2)+k*8] = alpha_sum[k];
                float rcp = 1.f/(alpha_sum[k]+1e-16f);
                alpha[k] *= rcp;
                alpha[2+k] *= rcp;
            }
            for (int k = 0; k < 4; k++) 
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[k]) : "f"(alpha[k]));
        }
        // Matmul
        {
            for (int k = 0; k < 2; k++) {
                float update = __expf(softmax[i&1][0][(lane_id>>2)+k*8]-softmax[(i+1)&1][0][(lane_id>>2)+k*8])*(softmax[i&1][1][(lane_id>>2)+k*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+k*8]+1e-16f);
                for (int j = 0; j < 4; j++) {
                    D[k*2+(j&1)+(j&2)*2] *= update;
                }
            }
            for (int k = 0; k < 2; k++) {
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                    : "=f"(D[k*4]), "=f"(D[k*4+1]), "=f"(D[k*4+2]), "=f"(D[k*4+3]) 
                    : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
                    "r"(B[k]), "r"(B[2+k])
                    "f"(D[k*4]), "f"(D[k*4+1]), "f"(D[k*4+2]), "f"(D[k*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    }
    i = block_end - 1;
    for (int k = 0; k < 4; k++) {
        if (SparseAToX_idx[i&1][(lane_id&3)+(k&2)*2] < node_num)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(dense[i&1][(lane_id&3)+(k&2)*2][warp_id*16+(k&1)*8+(lane_id>>2)]));
        else
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[k]) : "f"(0.f));
    }
    // softmax
    {
        int row_start = BitMaskRowOffset[i];
        int row_end = BitMaskRowOffset[i+1];
        float alpha[4]; 
        uint32_t col_mask = BitColMask[i];
        float mask[4], alpha_max[2], alpha_sum[2];
        for (int k = 0; k < 4; k++) {
            int row_id = (lane_id>>2)+(k&1)*8;
            int col_id = (lane_id&3)+(k&2)*2;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            mask[k] = (col_mask>>row_id)&(row_mask>>col_id)&1;
            if (mask[k])
                alpha[k] = leaky_relu(alphai[bid * 16 + row_id] + alphaj[SparseAToX_idx[i&1][col_id]]);
            else 
                alpha[k] = std::numeric_limits<float>::lowest();
        }
        for (int k = 0; k < 2; k++) {
            alpha_max[k] = max(alpha[k], alpha[2+k]);
            for (int j = 1; j < 4; j*=2) {
                alpha_max[k] = max(alpha_max[k], __shfl_xor_sync(FULL_MASK, alpha_max[k], 2*j-1, 4));
            }
            alpha_max[k] = max(alpha_max[k], softmax[i&1][0][(lane_id>>2)+k*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+k*8] = alpha_max[k];
            alpha[k] = mask[k] * __expf(alpha[k] - alpha_max[k]);
            alpha[2+k] = mask[2+k] * __expf(alpha[2+k] - alpha_max[k]);
            alpha_sum[k] = alpha[k] + alpha[2+k];
            for (int j = 1; j < 4; j*=2) {
                alpha_sum[k] += __shfl_xor_sync(FULL_MASK, alpha_sum[k], 2*j-1, 4);
            }
            alpha_sum[k] += softmax[i&1][1][(lane_id>>2)+k*8] * __expf(softmax[i&1][0][(lane_id>>2)+k*8] - alpha_max[k]);
            softmax[(i+1)&1][1][(lane_id>>2)+k*8] = alpha_sum[k];
            float rcp = 1.f/(alpha_sum[k]+1e-16f);
            alpha[k] *= rcp;
            alpha[2+k] *= rcp;
        }
        for (int k = 0; k < 4; k++) 
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[k]) : "f"(alpha[k]));
    }
    // Matmul
    {
        for (int k = 0; k < 2; k++) {
            float update = __expf(softmax[i&1][0][(lane_id>>2)+k*8]-softmax[(i+1)&1][0][(lane_id>>2)+k*8])*(softmax[i&1][1][(lane_id>>2)+k*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+k*8]+1e-16f);
            for (int j = 0; j < 4; j++) {
                D[k*2+(j&1)+(j&2)*2] *= update;
            }
        }
        for (int k = 0; k < 2; k++) {
            asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                : "=f"(D[k*4]), "=f"(D[k*4+1]), "=f"(D[k*4+2]), "=f"(D[k*4+3]) 
                : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
                "r"(B[k]), "r"(B[2+k])
                "f"(D[k*4]), "f"(D[k*4+1]), "f"(D[k*4+2]), "f"(D[k*4+3]));
        }
    }
    for (int k = 0; k < 4; k++) 
        *(float2*)(&out[(bid*16+(lane_id>>2)+(k&1)*8)*64+warp_id*16+(k&2)*4+(lane_id&3)*2]) = *(float2*)(&D[k*2]);
}

__global__ void gat_kernel_16x16_64_2_(
    const int* __restrict__ RowWindowOffsets,
    const int* __restrict__ SparseAToX,
    const int* __restrict__ BitMaskRowOffset,
    const uint16_t* __restrict__ BitColMask,
    const uint16_t* __restrict__ BitRowMask,
    const float* __restrict__ feat,
    const float* __restrict__ alphai,
    const float* __restrict__ alphaj,
    float* __restrict__ out,
    const int node_num
) {
    int bid = blockIdx.x;
    int lane_id = threadIdx.x & 31;
    int warp_id = threadIdx.x >> 5;
    int block_start = RowWindowOffsets[bid];
    int block_end = RowWindowOffsets[bid+1];

    if(block_end == block_start) return;

    __shared__ int SparseAToX_idx[2][16];
    __shared__ float dense[2][16][64];
    __shared__ float softmax[2][2][16];

    int i = block_start;
    // if (warp_id == 0) {
    softmax[i&1][0][lane_id&15] = std::numeric_limits<float>::lowest();
    softmax[i&1][1][lane_id&15] = 0.f;
    // }
    for (int j = 0; j < 2; j++) {
        SparseAToX_idx[i&1][(threadIdx.x>>4)+j*8] = SparseAToX[i*16+j*8+(threadIdx.x>>4)];
        int cur_addr = __cvta_generic_to_shared(&dense[i&1][j*8+(threadIdx.x>>4)][(lane_id&15)*4]);
        asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[i&1][j*8+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
    }
    asm volatile("cp.async.commit_group;\n"::);
    asm volatile("cp.async.wait_group 0;\n"::);
    __syncthreads();
    float D[8] = {0.f};
    uint32_t A[8], B[8];
    for (int i = block_start; i < (block_end - 1); i++) {
        // load and init
        for (int j = 0; j < 2; j++) {
            SparseAToX_idx[(i+1)&1][(threadIdx.x>>4)+j*8] = SparseAToX[(i+1)*16+8*j+(threadIdx.x>>4)];
            int cur_addr = __cvta_generic_to_shared(&dense[(i+1)&1][j*8+(threadIdx.x>>4)][(lane_id&15)*4]);
            asm volatile("cp.async.ca.shared.global [%0], [%1], 16;\n" : : "r"(cur_addr), "l"(&feat[min(SparseAToX_idx[(i+1)&1][j*8+(threadIdx.x>>4)], node_num-1)*64+(lane_id&15)*4]));
        }
        asm volatile("cp.async.commit_group;\n"::);
        for (int j = 0; j < 8; j++) {
            if (SparseAToX_idx[i&1][(lane_id&3)+(j&6)*2] < node_num)
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense[i&1][(lane_id&3)+(j&6)*2][warp_id*16+(j&1)*8+(lane_id>>2)]));
            else
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));
        }
        // softmax
        {
            int row_start = BitMaskRowOffset[i]/2;
            // int row_end = BitMaskRowOffset[i+1];
            float alpha[8];
            uint32_t col_mask = BitColMask[i];
            float mask[8], alpha_max[2], alpha_sum[2];
            for (int j = 0; j < 8; j++) {
                int row_id = (lane_id>>2)+(j&1)*8;
                int col_id = (lane_id&3)+(j&6)*2;
                uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
                mask[j] = (col_mask>>row_id)&(row_mask>>col_id)&1;
                if (mask[j])
                    alpha[j] = leaky_relu(alphai[min(bid*16+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[i&1][col_id], node_num-1)]);
                else
                    alpha[j] = std::numeric_limits<float>::lowest();
            }
            for (int j = 0; j < 2; j++) {
                alpha_max[j] = max(alpha[j], alpha[2+j]);
                alpha_max[j] = max(alpha_max[j], max(alpha[4+j], alpha[6+j]));
                for (int k = 1; k < 4; k*=2) {
                    alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
                }
                alpha_max[j] = max(alpha_max[j], softmax[i&1][0][(lane_id>>2)+j*8]);
                softmax[(i+1)&1][0][(lane_id>>2)+j*8] = alpha_max[j];
                alpha[j] = mask[j] * __expf(alpha[j] - alpha_max[j]);
                alpha[2+j] = mask[2+j] * __expf(alpha[2+j] - alpha_max[j]);
                alpha[4+j] = mask[4+j] * __expf(alpha[4+j] - alpha_max[j]);
                alpha[6+j] = mask[6+j] * __expf(alpha[6+j] - alpha_max[j]);
                alpha_sum[j] = alpha[j] + alpha[2+j] + alpha[4+j] + alpha[6+j];
                for (int k = 1; k < 4; k*=2) {
                    alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
                }
                alpha_sum[j] += softmax[i&1][1][(lane_id>>2)+j*8] * __expf(softmax[i&1][0][(lane_id>>2)+j*8] - alpha_max[j]);
                softmax[(i+1)&1][1][(lane_id>>2)+j*8] = alpha_sum[j];
                float rcp = 1.f/(alpha_sum[j]+1e-16f);
                alpha[j] *= rcp;
                alpha[2+j] *= rcp;
                alpha[4+j] *= rcp;
                alpha[6+j] *= rcp;
            }
            for (int j = 0; j < 8; j++) {
                asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
            }
        }
        // Matmul
        {
            for (int j = 0; j < 2; j++) {
                float update = __expf(softmax[i&1][0][(lane_id>>2)+j*8]-softmax[(i+1)&1][0][(lane_id>>2)+j*8])*(softmax[i&1][1][(lane_id>>2)+j*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+j*8]+1e-16f);
                for (int k = 0; k < 4; k++) {
                    D[j*2+(k&1)+(k&2)*2] *= update;
                }
            }
            for (int j = 0; j < 4; j++) {
                asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                    : "=f"(D[(j&1)*4]), "=f"(D[(j&1)*4+1]), "=f"(D[(j&1)*4+2]), "=f"(D[(j&1)*4+3]) 
                    : "r"(A[(j&2)*2]), "r"(A[(j&2)*2+1]), "r"(A[(j&2)*2+2]), "r"(A[(j&2)*2+3]), 
                    "r"(B[(j&2)*2+(j&1)]), "r"(B[2+(j&2)*2+(j&1)]),
                    "f"(D[(j&1)*4]), "f"(D[(j&1)*4+1]), "f"(D[(j&1)*4+2]), "f"(D[(j&1)*4+3]));
            }
        }
        asm volatile("cp.async.wait_group 0;\n"::);
        __syncthreads();
    } 
    i = block_end - 1;
    for (int j = 0; j < 8; j++) {
        if (SparseAToX_idx[i&1][(lane_id&3)+(j&6)*2] < node_num)
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(dense[i&1][(lane_id&3)+(j&6)*2][warp_id*16+(j&1)*8+(lane_id>>2)]));
        else
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(B[j]) : "f"(0.f));
    }
    // softmax
    {
        int row_start = BitMaskRowOffset[i]/2;
        // int row_end = BitMaskRowOffset[i+1];
        float alpha[8];
        uint32_t col_mask = BitColMask[i];
        float mask[8], alpha_max[2], alpha_sum[2];
        for (int j = 0; j < 8; j++) {
            int row_id = (lane_id>>2)+(j&1)*8;
            int col_id = (lane_id&3)+(j&6)*2;
            uint32_t row_mask = BitRowMask[row_start+__popc(col_mask&((1<<row_id)-1))];
            mask[j] = (col_mask>>row_id)&(row_mask>>col_id)&1;
            if (mask[j])
                alpha[j] = leaky_relu(alphai[min(bid*16+row_id, node_num-1)]+alphaj[min(SparseAToX_idx[i&1][col_id], node_num-1)]);
            else
                alpha[j] = std::numeric_limits<float>::lowest();
        }
        for (int j = 0; j < 2; j++) {
            alpha_max[j] = max(alpha[j], alpha[2+j]);
            alpha_max[j] = max(alpha_max[j], max(alpha[4+j], alpha[6+j]));
            for (int k = 1; k < 4; k*=2) {
                alpha_max[j] = max(alpha_max[j], __shfl_xor_sync(FULL_MASK, alpha_max[j], 2*k-1, 4));
            }
            alpha_max[j] = max(alpha_max[j], softmax[i&1][0][(lane_id>>2)+j*8]);
            softmax[(i+1)&1][0][(lane_id>>2)+j*8] = alpha_max[j];
            alpha[j] = mask[j] * __expf(alpha[j] - alpha_max[j]);
            alpha[2+j] = mask[2+j] * __expf(alpha[2+j] - alpha_max[j]);
            alpha[4+j] = mask[4+j] * __expf(alpha[4+j] - alpha_max[j]);
            alpha[6+j] = mask[6+j] * __expf(alpha[6+j] - alpha_max[j]);
            alpha_sum[j] = alpha[j] + alpha[2+j] + alpha[4+j] + alpha[6+j];
            for (int k = 1; k < 4; k*=2) {
                alpha_sum[j] += __shfl_xor_sync(FULL_MASK, alpha_sum[j], 2*k-1, 4);
            }
            alpha_sum[j] += softmax[i&1][1][(lane_id>>2)+j*8] * __expf(softmax[i&1][0][(lane_id>>2)+j*8] - alpha_max[j]);
            softmax[(i+1)&1][1][(lane_id>>2)+j*8] = alpha_sum[j];
            float rcp = 1.f/(alpha_sum[j]+1e-16f);
            alpha[j] *= rcp;
            alpha[2+j] *= rcp;
            alpha[4+j] *= rcp;
            alpha[6+j] *= rcp;
        }
        for (int j = 0; j < 8; j++) {
            asm volatile("cvt.rna.tf32.f32 %0, %1;\n" : "=r"(A[j]) : "f"(alpha[j]));
        }
    }
    // Matmul
    {
        for (int j = 0; j < 2; j++) {
            float update = __expf(softmax[i&1][0][(lane_id>>2)+j*8]-softmax[(i+1)&1][0][(lane_id>>2)+j*8])*(softmax[i&1][1][(lane_id>>2)+j*8]+1e-16f)/(softmax[(i+1)&1][1][(lane_id>>2)+j*8]+1e-16f);
            for (int k = 0; k < 4; k++) {
                D[j*2+(k&1)+(k&2)*2] *= update;
            }
        }
        for (int j = 0; j < 4; j++) {
            asm volatile("mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};\n" 
                : "=f"(D[(j&1)*4]), "=f"(D[(j&1)*4+1]), "=f"(D[(j&1)*4+2]), "=f"(D[(j&1)*4+3]) 
                : "r"(A[(j&2)*2]), "r"(A[(j&2)*2+1]), "r"(A[(j&2)*2+2]), "r"(A[(j&2)*2+3]), 
                "r"(B[(j&2)*2+(j&1)]), "r"(B[2+(j&2)*2+(j&1)]),
                "f"(D[(j&1)*4]), "f"(D[(j&1)*4+1]), "f"(D[(j&1)*4+2]), "f"(D[(j&1)*4+3]));
        }
    }
    for (int j = 0; j < 4; j++) 
        if ((bid*16+(lane_id>>2)+(j&1)*8)<node_num)
            *(float2*)(&out[(bid*16+(lane_id>>2)+(j&1)*8)*64+warp_id*16+(j&2)*4+(lane_id&3)*2]) = *(float2*)(&D[j*2]);
}

at::Tensor GAT_adaptive(
    at::Tensor feature,
    at::Tensor lin_weight,
    at::Tensor att_i,
    at::Tensor att_j,
    at::Tensor RowWindowId_8x8,
    at::Tensor RowWindowId_8x16,
    at::Tensor RowWindowId_16x8,
    at::Tensor RowWindowId_16x16,
    at::Tensor RowWindowRowOffset,
    at::Tensor RowWindowBlockOffset,
    at::Tensor RowWindowSparseAToXOffset,
    at::Tensor RowWindowColMaskOffset,
    at::Tensor BlockRowMaskOffset,
    at::Tensor ColMask,
    at::Tensor RowMask,
    at::Tensor SparseAToX
) {
    int num_nodes = feature.size(0);
    auto feats = torch::mm(feature, lin_weight.t());
    auto alpha_i = torch::matmul(feats, att_i.squeeze());
    auto alpha_j = torch::matmul(feats, att_j.squeeze());
    int out_feats = feats.size(1);
    auto output = at::empty({num_nodes, out_feats}, feature.options()).fill_(0);
    int block_8x8 = RowWindowId_8x8.size(0);
    int block_8x16 = RowWindowId_8x16.size(0);
    int block_16x8 = RowWindowId_16x8.size(0);
    int block_16x16 = RowWindowId_16x16.size(0);
    uint8_t select = (block_8x8>0)+(block_8x16>0)*(1<<1)+(block_16x8>0)*(1<<2)+(block_16x16>0)*(1<<3);
    hipStream_t stream[4];
    hipStreamCreateWithFlags(&stream[0], hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream[1], hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream[2], hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream[3], hipStreamNonBlocking);
    switch(select) {
        case 1:
            gat_kernel_8x8_64_2_<<<block_8x8, 64>>>(
                RowWindowBlockOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 2:
            gat_kernel_8x16_64_2_<<<block_8x16, 64>>>(
                RowWindowBlockOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                (uint16_t*)RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 4:
            gat_kernel_16x8_64_2_<<<block_16x8, 128>>>(
                RowWindowBlockOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                (uint16_t*)ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 8:
            gat_kernel_16x16_64_2_<<<block_16x16, 128>>>(
                RowWindowBlockOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                (uint16_t*)ColMask.data_ptr<uint8_t>(),
                (uint16_t*)RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 3:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_8x8_64<<<block_8x16, 64, 0, stream[1]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 5:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[1]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 9:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[1]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 6:
            gat_kernel_8x16_64<<<block_8x16, 64, 0, stream[0]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[1]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 10:
            gat_kernel_8x16_64<<<block_8x16, 64, 0, stream[0]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[1]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 12:
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[0]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[1]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 7:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_8x16_64<<<block_8x16, 64, 0, stream[1]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[2]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 11:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_8x16_64<<<block_8x16, 64, 0, stream[1]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[2]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 13:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[1]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[2]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 14:
            gat_kernel_8x16_64<<<block_8x16, 64, 0, stream[0]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[1]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[2]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        case 15:
            gat_kernel_8x8_64<<<block_8x8, 64, 0, stream[0]>>>(
                RowWindowId_8x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_8x16_64<<<block_8x16, 64, 0, stream[1]>>>(
                RowWindowId_8x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x8_64<<<block_16x8, 128, 0, stream[2]>>>(
                RowWindowId_16x8.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            gat_kernel_16x16_64<<<block_16x16, 128, 0, stream[3]>>>(
                RowWindowId_16x16.data_ptr<int>(),
                RowWindowRowOffset.data_ptr<int>(),
                RowWindowBlockOffset.data_ptr<int>(),
                RowWindowSparseAToXOffset.data_ptr<int>(),
                RowWindowColMaskOffset.data_ptr<int>(),
                SparseAToX.data_ptr<int>(),
                BlockRowMaskOffset.data_ptr<int>(),
                ColMask.data_ptr<uint8_t>(),
                RowMask.data_ptr<uint8_t>(),
                feats.data_ptr<float>(),
                alpha_i.data_ptr<float>(),
                alpha_j.data_ptr<float>(),
                output.data_ptr<float>(),
                num_nodes);
            break;
        default:
            printf("Unsupported select: %d\n", select);
            exit(1);
    }
    return output;
}