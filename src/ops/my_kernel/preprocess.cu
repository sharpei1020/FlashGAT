#include "hip/hip_runtime.h"
#include "preprocess.cuh"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/gather.h>
#include <thrust/adjacent_difference.h>
#include <thrust/transform_scan.h>
#include <omp.h>



std::vector<at::Tensor> preprocess_CSR(at::Tensor edge_index, at::Tensor counts, int group, int num_nodes)
{
    auto offset = at::empty({num_nodes}, device(torch::kCUDA)).to(torch::kInt);
    thrust::inclusive_scan(thrust::device, counts.data_ptr<int>(), counts.data_ptr<int>() + num_nodes, offset.data_ptr<int>());
    int num_edges = edge_index.size(1);
    auto out_edge_index = at::empty({num_edges}, device(torch::kCUDA)).to(torch::kInt);
    thrust::sequence(thrust::device, out_edge_index.data_ptr<int>(), out_edge_index.data_ptr<int>() + num_edges);
    u_int32_t* value;
    hipMalloc(&value, num_edges * sizeof(u_int32_t));
    thrust::transform(thrust::device, edge_index.data_ptr<int>(), 
                      edge_index.data_ptr<int>() + num_edges, edge_index.data_ptr<int>() + num_edges,
                      value, [=] __device__ (int e0, int e1) {return ((uint32_t)(e1 / group) * (uint32_t)num_nodes + (uint32_t)e0);});
    thrust::stable_sort_by_key(thrust::device, value, value + num_edges, out_edge_index.data_ptr<int>());
    hipFree(value);
    return {offset, out_edge_index};
}

std::vector<at::Tensor> get_graph_set(at::Tensor edge_index, int numwarps, int numnodes)
{
    int num_edges = edge_index.size(1);
    auto value = at::empty({num_edges}, device(torch::kCUDA)).to(torch::kInt64);
    thrust::transform(thrust::device, edge_index.data_ptr<int>(), 
                      edge_index.data_ptr<int>() + num_edges, edge_index.data_ptr<int>() + num_edges,
                      value.data_ptr<int64_t>(), [=] __device__ (int e0, int e1) {return (int64_t)e1 * (int64_t)numnodes + (int64_t)e0;});             
    int groups = (numnodes + numwarps - 1) / numwarps;
    std::vector<at::Tensor> graph_set(groups), graph_mask(groups);
    std::vector<int64_t> graph_size(groups + 1);
    graph_size[0] = 0;
    #pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < groups; i++) {
        graph_set[i] = edge_index.slice(0, 0, 1).masked_select(value >= (int64_t)i * (int64_t)numwarps * (int64_t)numnodes & value < (int64_t)(i + 1) * (int64_t)numwarps * (int64_t)numnodes);
        graph_set[i] = std::get<0>(torch::_unique(graph_set[i]));
        // printf("graph_set[%d] size: %d\n", i, graph_set[i].size(0));
        graph_size[i + 1] = graph_set[i].size(0);
        // auto new_end = thrust::unique(thrust::device, graph_set[i].data_ptr<int>(), graph_set[i].data_ptr<int>() + graph_set[i].size(0));
        // graph_size[i + 1] = new_end - graph_set[i].data_ptr<int>();
        graph_mask[i] = at::empty({numwarps, graph_size[i + 1]}, device(torch::kCUDA)).to(torch::kBool);
        for (int j = 0; j < numwarps; j++) {
            auto temp_mask = at::empty({numnodes}, device(torch::kCUDA)).to(torch::kBool).fill_(false);
            auto temp_idx = edge_index.slice(0, 0, 1).masked_select(value >= (int64_t)(i * numwarps + j) * (int64_t)numnodes & value < (int64_t)(i * numwarps + j + 1) * (int64_t)numnodes).to(torch::kInt64);
            temp_mask.index_fill_(0, temp_idx, true); 
            // printf("%d \n", graph_size[i + 1]);
            thrust::gather(thrust::device, graph_set[i].data_ptr<int>(), graph_set[i].data_ptr<int>() + graph_size[i + 1], 
                            temp_mask.data_ptr<bool>(), graph_mask[i].data_ptr<bool>() + j * graph_size[i + 1]);
        }
    }
    printf("get_graph_set done\n");
    for (int i = 0; i < groups; i++) 
        graph_size[i + 1] += graph_size[i];
    auto graphset = at::empty({graph_size[groups]}, device(torch::kCUDA)).to(torch::kInt);
    auto graphmask = at::empty({graph_size[groups], numwarps}, device(torch::kCUDA)).to(torch::kBool);
    auto graphoffset = at::empty({groups + 1}, device(torch::kCUDA)).to(torch::kInt64);
    hipMemcpy(graphoffset.data_ptr<int64_t>(), &graph_size[0], sizeof(int64_t), hipMemcpyHostToDevice);
    #pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < groups; i++) {
        auto temp_mask = graph_mask[i].transpose(0, 1).contiguous();
        thrust::copy(thrust::device, graph_set[i].data_ptr<int>(), graph_set[i].data_ptr<int>() + graph_set[i].size(0), 
                    graphset.data_ptr<int>() + graph_size[i]);
        thrust::copy(thrust::device, temp_mask.data_ptr<bool>(), temp_mask.data_ptr<bool>() + temp_mask.size(0) * temp_mask.size(1), 
                    graphmask.data_ptr<bool>() + graph_size[i] * numwarps);
        hipMemcpy(graphoffset.data_ptr<int64_t>() + i + 1, &graph_size[i + 1], sizeof(int64_t), hipMemcpyHostToDevice);
    }
    return {graphset, graphmask, graphoffset};
}

__global__ void BlockMaskKernel(
    int* edge_index_0,
    int* edge_index_1,
    u_int32_t* TCOffset,
    uint8_t* BlockMask,
    int block_high,
    int block_width,
    int block_num
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= block_num) return;
    u_int32_t start = TCOffset[tid];
    u_int32_t end = TCOffset[tid + 1];
    uint8_t mask[16] = {0};
     
    for (uint32_t i = start; i < end; i++) {
        int block_col = ((edge_index_0[i] - 1) % block_width);
        int block_row = (edge_index_1[i] % block_high);
        mask[block_row] += (uint8_t)(1 << block_col);
    }
    *((int4 *)(&BlockMask[tid * 16])) = *((int4 *)(&mask[0]));
}

std::vector<torch::Tensor> process_DTC(at::Tensor edge_index, at::Tensor dev_idx, int block_high, int block_width, int num_nodes)
{
    uint32_t* value, *value1;
    // , *value2
    int* rowwindow_value, *X_col_id, *edge_idx1;
    hipMalloc(&value, edge_index.size(1) * sizeof(uint32_t));
    hipMalloc(&value1, edge_index.size(1) * sizeof(uint32_t));
    hipMalloc(&edge_idx1, edge_index.size(1) * sizeof(int));
    thrust::gather(thrust::device, edge_index.data_ptr<int>() + edge_index.size(1), edge_index.data_ptr<int>() + 2 * edge_index.size(1), dev_idx.data_ptr<int>(), edge_idx1);
    // hipMalloc(&value2, edge_index.size(1) * sizeof(uint32_t));
    thrust::transform(thrust::device, edge_index.data_ptr<int>(), 
                      edge_index.data_ptr<int>() + edge_index.size(1), edge_idx1,
                      value, [=] __device__ (int e0, int e1) {return ((uint32_t)(e1 / block_high) * (uint32_t)(num_nodes) + (uint32_t)(e0));});
    thrust::copy(thrust::device, value, value + edge_index.size(1), value1);
    // thrust::copy(thrust::device, value, value + edge_index.size(1), value2);
    
    thrust::stable_sort_by_key(thrust::device, value, value + edge_index.size(1), edge_index.data_ptr<int>());
    thrust::stable_sort_by_key(thrust::device, value1, value1 + edge_index.size(1), edge_idx1);
    // thrust::stable_sort_by_key(thrust::device, value2, value + edge_index.size(1), rowwindow_value);
    hipFree(value);
    hipFree(value1);
    hipMalloc(&rowwindow_value, edge_index.size(1) * sizeof(int));

    thrust::transform(thrust::device, edge_idx1, edge_idx1 + edge_index.size(1),
                      rowwindow_value, [=] __device__ (int e) {return e / block_high;});

    int* SparseAidx;
    hipMalloc(&SparseAidx, edge_index.size(1) * sizeof(int));
    thrust::adjacent_difference(thrust::device, edge_index.data_ptr<int>(), edge_index.data_ptr<int>() + edge_index.size(1), 
                                SparseAidx, [=]__device__(int x1, int x0) {return (x1 == x0) ? 0 : 1;});
    hipMemset(SparseAidx, 1, sizeof(int));
    thrust::inclusive_scan_by_key(thrust::device, rowwindow_value, rowwindow_value + edge_index.size(1), SparseAidx, SparseAidx, 
                                thrust::equal_to<int>(), thrust::plus<int>());

    hipMalloc(&X_col_id, edge_index.size(1) * sizeof(int));
    thrust::copy(thrust::device, edge_index.data_ptr<int>(), edge_index.data_ptr<int>() + edge_index.size(1), X_col_id);
    auto unique_rowwindow_end = thrust::unique_by_key(thrust::device, X_col_id, X_col_id + edge_index.size(1), 
                            rowwindow_value, thrust::equal_to<int>());
    //SparseAToX
    auto SparseAToX = torch::from_blob(X_col_id, {unique_rowwindow_end.first - X_col_id}, device(torch::kCUDA).dtype(torch::kInt32)).clone();
    int* RowWindowMask, *RowWindowNum_temp, *RowWindowNum, *unique_rowwindow_value;
    hipMalloc(&RowWindowMask, SparseAToX.size(0) * sizeof(int));
    hipMalloc(&RowWindowNum_temp, SparseAToX.size(0) * sizeof(int));
    hipMalloc(&RowWindowNum, (num_nodes + block_high - 1) / block_high * sizeof(int));
    thrust::fill_n(thrust::device, RowWindowNum, (num_nodes + block_high - 1) / block_high, 0);
    hipMalloc(&unique_rowwindow_value, SparseAToX.size(0) * sizeof(int));
    thrust::fill_n(thrust::device, RowWindowMask, SparseAToX.size(0), 1);
    // printf("SparseAToX size: %d\n", SparseAToX.size(0));
    auto RowWindowColOffset = at::empty({1 + (num_nodes + block_high - 1) / block_high}, device(torch::kCUDA).dtype(torch::kInt64)).fill_(0);
    auto new_end_ = thrust::reduce_by_key(thrust::device, rowwindow_value, unique_rowwindow_end.second, RowWindowMask,
                            unique_rowwindow_value, RowWindowNum_temp, thrust::equal_to<int>(), thrust::plus<int>());
    thrust::scatter(thrust::device, RowWindowNum_temp, new_end_.second, unique_rowwindow_value, RowWindowNum);
    thrust::inclusive_scan(thrust::device, RowWindowNum, RowWindowNum + (num_nodes + block_high - 1) / block_high, RowWindowColOffset.data_ptr<int64_t>() + 1);
    thrust::transform(thrust::device, RowWindowNum, RowWindowNum + (num_nodes + block_high - 1) / block_high, RowWindowNum, [=]__device__(int x) {return (x + block_width - 1) / block_width;});
    auto RowWindowOffset = at::empty({(num_nodes + block_high - 1) / block_high + 1}, device(torch::kCUDA).dtype(torch::kInt32)).fill_(0);
    //RowWindowOffset
    thrust::inclusive_scan(thrust::device, RowWindowNum, RowWindowNum + (num_nodes + block_high - 1) / block_high, RowWindowOffset.data_ptr<int>() + 1);
    hipFree(X_col_id);
    hipFree(RowWindowMask);
    hipFree(RowWindowNum_temp);
    hipFree(RowWindowNum);
    hipFree(unique_rowwindow_value);
    
    int block_num = RowWindowOffset[-1].item().to<int>();
    // printf("block_num: %d\n", block_num);
    // auto TCOffset = at::empty({block_num + 1}, device(torch::kCUDA).dtype(torch::kInt64)).fill_(0);
    uint32_t *block_value, *TCOffset, *BlockElementMask, *BlockElementNum;
    hipMalloc(&TCOffset, (block_num + 1) * sizeof(uint32_t));
    hipMalloc(&block_value, edge_index.size(1) * sizeof(uint32_t));
    hipMalloc(&BlockElementMask, edge_index.size(1) * sizeof(uint32_t));
    hipMalloc(&BlockElementNum, block_num * sizeof(uint32_t));
    hipMemset(TCOffset, 0, sizeof(uint32_t));
    thrust::fill_n(thrust::device, BlockElementMask, edge_index.size(1), 1);
    thrust::transform(thrust::device, SparseAidx, SparseAidx + edge_index.size(1), edge_idx1,
                    block_value, 
                    [=] __device__ (int e0, int e1) {return ((uint32_t)(e1 / block_high) * (uint32_t)(num_nodes) + (uint32_t)((e0-1) / block_width));});
    auto new_end = thrust::reduce_by_key(thrust::device, block_value, block_value + (int)edge_index.size(1), BlockElementMask,
                            block_value, BlockElementNum, thrust::equal_to<uint32_t>(), thrust::plus<uint32_t>());
    thrust::inclusive_scan(thrust::device, BlockElementNum, BlockElementNum + block_num, TCOffset + 1);
    
    //BlockMask
    auto BlockMask = at::empty({16 * block_num}, device(torch::kCUDA).dtype(torch::kUInt8));
    int grid = (block_num + 127) / 128;
    BlockMaskKernel<<<grid, 128>>>(SparseAidx, 
                                    edge_idx1, 
                                    TCOffset, 
                                    BlockMask.data_ptr<uint8_t>(), 
                                    block_high, block_width, block_num);
    hipFree(rowwindow_value);
    hipFree(SparseAidx);
    hipFree(TCOffset);
    hipFree(block_value);
    hipFree(BlockElementMask);
    hipFree(BlockElementNum);
    return {RowWindowOffset, RowWindowColOffset, BlockMask, SparseAToX};
}
    